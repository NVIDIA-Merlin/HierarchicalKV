#include <hip/hip_runtime.h>

#include <algorithm>
#include <chrono>
#include <iostream>
#include <random>
#include <thread>
#include <unordered_set>

typedef float V;

constexpr int DIM = 64;
struct Vector {
  V values[DIM];
};

void create_random_offset(int *offset, int num, int range) {
  std::unordered_set<int> numbers;
  std::random_device rd;
  std::mt19937_64 eng(rd());
  std::uniform_int_distribution<int> distr;
  int i = 0;

  while (numbers.size() < num) {
    numbers.insert(distr(eng) % range);
  }

  for (const int num : numbers) {
    offset[i++] = num;
  }
}

__global__ void d2h_const_data(const Vector *__restrict src,
                               Vector **__restrict dst, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    int vec_index = int(tid / DIM);
    int dim_index = tid % DIM;

    (*(dst[vec_index])).values[dim_index] = 0.1f;
  }
}

__global__ void d2h_hbm_data(
    Vector *__restrict src, Vector **__restrict dst,
    int N) {  // dst is a set of Vector* in the pinned memory
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    int vec_index = int(tid / DIM);
    int dim_index = tid % DIM;

    //     (*(dst[vec_index])).values[dim_index] =
    //     src[vec_index].values[dim_index];

    src[vec_index].values[dim_index] = (*(dst[vec_index])).values[dim_index];
  }
}

__global__ void create_fake_ptr(const Vector *__restrict dst,
                                Vector **__restrict vectors, int *offset,
                                int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    vectors[tid] = (Vector *)((Vector *)dst + offset[tid]);
  }
}

int main() {
  constexpr int KEY_NUM = 1024 * 1024;
  constexpr int INIT_SIZE = KEY_NUM * 32;
  constexpr int N = KEY_NUM * DIM;
  constexpr int TEST_TIMES = 1;
  constexpr size_t vectors_size = INIT_SIZE * sizeof(Vector);

  int NUM_THREADS = 1024;
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  int *h_offset;
  int *d_offset;

  hipHostMalloc(&h_offset, sizeof(int) * KEY_NUM, hipHostMallocDefault);
  hipMalloc(&d_offset, sizeof(int) * KEY_NUM);
  hipMemset(&h_offset, 0, sizeof(int) * KEY_NUM);
  hipMemset(&d_offset, 0, sizeof(int) * KEY_NUM);

  Vector *src;
  Vector *dst;
  Vector **dst_ptr;
  hipMalloc(&src, KEY_NUM * sizeof(Vector));
  hipMalloc(&dst_ptr, KEY_NUM * sizeof(Vector *));
  hipHostMalloc(&dst, vectors_size, hipHostMallocDefault);

  create_random_offset(h_offset, KEY_NUM, INIT_SIZE);
  hipMemcpy(d_offset, h_offset, sizeof(int) * KEY_NUM, hipMemcpyHostToDevice);
  create_fake_ptr<<<1024, 1024>>>(dst, dst_ptr, d_offset, KEY_NUM);
  std::chrono::time_point<std::chrono::steady_clock> start_test;
  std::chrono::duration<double> diff_test;

  hipDeviceSynchronize();
  start_test = std::chrono::steady_clock::now();
  for (int i = 0; i < TEST_TIMES; i++) {
    d2h_const_data<<<NUM_BLOCKS, NUM_THREADS>>>(src, dst_ptr, N);
  }
  hipDeviceSynchronize();
  diff_test = std::chrono::steady_clock::now() - start_test;
  printf("[timing] Constant d2h=%.2fms\n",
         diff_test.count() * 1000 / TEST_TIMES);

  start_test = std::chrono::steady_clock::now();
  for (int i = 0; i < TEST_TIMES; i++) {
    d2h_hbm_data<<<NUM_BLOCKS, NUM_THREADS>>>(src, dst_ptr, N);
  }
  hipDeviceSynchronize();
  diff_test = std::chrono::steady_clock::now() - start_test;
  printf("[timing] HBM data d2h=%.2fms\n",
         diff_test.count() * 1000 / TEST_TIMES);

  hipHostFree(dst);
  hipHostFree(h_offset);
  hipFree(dst_ptr);
  hipFree(src);
  hipFree(d_offset);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
