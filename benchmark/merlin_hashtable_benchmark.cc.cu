#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <limits>
#include <random>
#include <thread>
#include <unordered_map>
#include <unordered_set>
#include "benchmark_util.cuh"
#include "merlin_hashtable.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::fixed;
using std::setfill;
using std::setprecision;
using std::setw;

using namespace nv::merlin;
using namespace benchmark;

enum class Test_Mode {
  pure_hbm = 0,
  hybrid = 1,
};

const float EPSILON = 0.001f;

std::string rep(int n) { return std::string(n, ' '); }

float test_one_api(const API_Select api, const size_t dim,
                   const size_t init_capacity, const size_t key_num_per_op,
                   const size_t hbm4values, const float load_factor,
                   const float hitrate = 0.6f, const bool io_by_cpu = false) {
  using K = uint64_t;
  using S = uint64_t;
  using V = float;
  using Table = nv::merlin::HashTable<K, float, S>;
  using TableOptions = nv::merlin::HashTableOptions;

  size_t free, total;
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMemGetInfo(&free, &total));

  if (free / (1 << 30) < hbm4values) {
    return 0.0f;
  }

  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_found;

  TableOptions options;

  options.init_capacity = init_capacity;
  options.max_capacity = init_capacity;
  options.dim = dim;
  options.max_hbm_for_vectors = nv::merlin::GB(hbm4values);
  options.io_by_cpu = io_by_cpu;
  options.evict_strategy = EvictStrategy::kLru;

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(options);

  CUDA_CHECK(hipHostMalloc(&h_keys, key_num_per_op * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, key_num_per_op * sizeof(S)));
  CUDA_CHECK(
      hipHostMalloc(&h_vectors, key_num_per_op * sizeof(V) * options.dim));
  CUDA_CHECK(hipHostMalloc(&h_found, key_num_per_op * sizeof(bool)));

  CUDA_CHECK(
      hipMemset(h_vectors, 0, key_num_per_op * sizeof(V) * options.dim));

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  V* d_def_val;
  V** d_vectors_ptr;
  bool* d_found;
  K* d_keys_out;

  K* d_evict_keys;
  S* d_evict_scores;

  CUDA_CHECK(hipMalloc(&d_keys, key_num_per_op * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_vectors, key_num_per_op * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_def_val, key_num_per_op * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_vectors_ptr, key_num_per_op * sizeof(V*)));
  CUDA_CHECK(hipMalloc(&d_found, key_num_per_op * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_keys_out, key_num_per_op * sizeof(K)));

  CUDA_CHECK(hipMalloc(&d_evict_keys, key_num_per_op * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_evict_scores, key_num_per_op * sizeof(S)));

  CUDA_CHECK(
      hipMemset(d_vectors, 1, key_num_per_op * sizeof(V) * options.dim));
  CUDA_CHECK(
      hipMemset(d_def_val, 2, key_num_per_op * sizeof(V) * options.dim));
  CUDA_CHECK(hipMemset(d_vectors_ptr, 0, key_num_per_op * sizeof(V*)));
  CUDA_CHECK(hipMemset(d_found, 0, key_num_per_op * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // initialize insert
  // step 1, no need to load load_factor
  uint64_t key_num_init = static_cast<uint64_t>(init_capacity * load_factor);
  const float target_load_factor = key_num_init * 1.0f / init_capacity;
  uint64_t key_num_remain = key_num_init % key_num_per_op == 0
                                ? key_num_per_op
                                : key_num_init % key_num_per_op;
  int32_t loop_num_init = (key_num_init + key_num_per_op - 1) / key_num_per_op;

  K start = 0UL;
  for (int i = 0; i < loop_num_init; i++) {
    uint64_t key_num_cur_insert =
        i == loop_num_init - 1 ? key_num_remain : key_num_per_op;
    create_continuous_keys<K, S>(h_keys, h_scores, key_num_cur_insert, start);
    CUDA_CHECK(hipMemcpy(d_keys, h_keys, key_num_cur_insert * sizeof(K),
                          hipMemcpyHostToDevice));
    table->insert_or_assign(key_num_cur_insert, d_keys, d_vectors, d_scores,
                            stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    start += key_num_cur_insert;
  }
  // step 2
  float real_load_factor = table->load_factor(stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  while (target_load_factor - real_load_factor > EPSILON) {
    auto key_num_append = static_cast<int64_t>(
        (target_load_factor - real_load_factor) * init_capacity);
    if (key_num_append <= 0) break;
    key_num_append =
        std::min(static_cast<int64_t>(key_num_per_op), key_num_append);
    create_continuous_keys<K, S>(h_keys, h_scores, key_num_append, start);
    CUDA_CHECK(hipMemcpy(d_keys, h_keys, key_num_append * sizeof(K),
                          hipMemcpyHostToDevice));
    table->insert_or_assign(key_num_append, d_keys, d_vectors, d_scores,
                            stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    start += key_num_append;
    real_load_factor = table->load_factor(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  // For trigger the kernel selection in advance.
  int key_num_per_op_warmup = 1;
  for (int i = 0; i < 9; i++) {
    switch (api) {
      case API_Select::find: {
        table->find(key_num_per_op_warmup, d_keys, d_vectors, d_found, d_scores,
                    stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        break;
      }
      case API_Select::insert_or_assign: {
        table->insert_or_assign(key_num_per_op_warmup, d_keys, d_vectors,
                                d_scores, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        break;
      }
      case API_Select::find_or_insert: {
        table->find_or_insert(key_num_per_op_warmup, d_keys, d_vectors,
                              d_scores, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        break;
      }
      case API_Select::assign: {
        table->assign(key_num_per_op_warmup, d_keys, d_def_val, d_scores,
                      stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        break;
      }
      case API_Select::insert_and_evict: {
        table->insert_and_evict(key_num_per_op_warmup, d_keys, d_vectors,
                                d_scores, d_evict_keys, d_def_val,
                                d_evict_scores, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        break;
      }
      case API_Select::find_ptr: {
        V** d_vectors_ptr = nullptr;
        CUDA_CHECK(
            hipMalloc(&d_vectors_ptr, key_num_per_op_warmup * sizeof(V*)));
        benchmark::array2ptr(d_vectors_ptr, d_vectors, options.dim,
                             key_num_per_op_warmup, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));
        table->find(1, d_keys, d_vectors_ptr, d_found, d_scores, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        benchmark::read_from_ptr(d_vectors_ptr, d_vectors, options.dim,
                                 key_num_per_op_warmup, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipFree(d_vectors_ptr));
        break;
      }
      case API_Select::find_or_insert_ptr: {
        V** d_vectors_ptr = nullptr;
        bool* d_found;
        CUDA_CHECK(hipMalloc(&d_found, key_num_per_op_warmup * sizeof(bool)));
        CUDA_CHECK(
            hipMalloc(&d_vectors_ptr, key_num_per_op_warmup * sizeof(V*)));
        benchmark::array2ptr(d_vectors_ptr, d_vectors, options.dim,
                             key_num_per_op_warmup, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        table->find_or_insert(key_num_per_op_warmup, d_keys, d_vectors_ptr,
                              d_found, d_scores, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipFree(d_vectors_ptr));
        CUDA_CHECK(hipFree(d_found));
        break;
      }
      default: {
        std::cout << "[Unsupport API]\n";
      }
    }
  }
  create_keys_for_hitrate<K, S>(h_keys, h_scores, key_num_per_op, hitrate,
                                Hit_Mode::last_insert, start, true /*reset*/);
  CUDA_CHECK(hipMemcpy(d_keys, h_keys, key_num_per_op * sizeof(K),
                        hipMemcpyHostToDevice));
  auto timer = benchmark::Timer<double>();
  switch (api) {
    case API_Select::find: {
      timer.start();
      table->find(key_num_per_op, d_keys, d_vectors, d_found, d_scores, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      break;
    }
    case API_Select::insert_or_assign: {
      timer.start();
      table->insert_or_assign(key_num_per_op, d_keys, d_vectors, d_scores,
                              stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      break;
    }
    case API_Select::find_or_insert: {
      timer.start();
      table->find_or_insert(key_num_per_op, d_keys, d_vectors, d_scores,
                            stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      break;
    }
    case API_Select::assign: {
      timer.start();
      table->assign(key_num_per_op, d_keys, d_def_val, d_scores, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      break;
    }
    case API_Select::insert_and_evict: {
      timer.start();
      table->insert_and_evict(key_num_per_op, d_keys, d_vectors, d_scores,
                              d_evict_keys, d_def_val, d_evict_scores, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      break;
    }
    case API_Select::find_ptr: {
      V** d_vectors_ptr = nullptr;
      CUDA_CHECK(hipMalloc(&d_vectors_ptr, key_num_per_op * sizeof(V*)));
      benchmark::array2ptr(d_vectors_ptr, d_vectors, options.dim,
                           key_num_per_op, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.start();
      table->find(key_num_per_op, d_keys, d_vectors_ptr, d_found, d_scores,
                  stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      benchmark::read_from_ptr(d_vectors_ptr, d_vectors, options.dim,
                               key_num_per_op, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      CUDA_CHECK(hipFree(d_vectors_ptr));
      break;
    }
    case API_Select::find_or_insert_ptr: {
      V** d_vectors_ptr = nullptr;
      bool* d_found;
      CUDA_CHECK(hipMalloc(&d_found, key_num_per_op * sizeof(bool)));
      CUDA_CHECK(hipMalloc(&d_vectors_ptr, key_num_per_op * sizeof(V*)));
      benchmark::array2ptr(d_vectors_ptr, d_vectors, options.dim,
                           key_num_per_op, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.start();
      table->find_or_insert(key_num_per_op, d_keys, d_vectors_ptr, d_found,
                            d_scores, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      CUDA_CHECK(hipFree(d_vectors_ptr));
      CUDA_CHECK(hipFree(d_found));
      break;
    }
    default: {
      std::cout << "[Unsupport API]\n";
    }
  }

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_found));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_def_val));
  CUDA_CHECK(hipFree(d_vectors_ptr));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipFree(d_evict_keys));
  CUDA_CHECK(hipFree(d_evict_scores));

  CUDA_CHECK(hipDeviceSynchronize());
  CudaCheckError();

  float througput =
      key_num_per_op / timer.getResult() / (1024 * 1024 * 1024.0f);
  return througput;
}

static Test_Mode test_mode = Test_Mode::pure_hbm;

void print_title() {
  cout << endl
       << "|    λ "
       << "| insert_or_assign "
       << "|   find "
       << "| find_or_insert "
       << "| assign "
       << "|  find* "
       << "| find_or_insert* ";
  if (Test_Mode::pure_hbm == test_mode) {
    cout << "| insert_and_evict ";
  }
  cout << "|\n";

  //<< "| load_factor "
  cout << "|-----:"
       //<< "| insert_or_assign "
       << "|-----------------:"
       //<< "|   find "
       << "|-------:"
       //<< "| find_or_insert "
       << "|---------------:"
       //<< "| assign "
       << "|-------:"
       //<< "|   find* "
       << "|-------:"
       //<< "| find_or_insert* "
       << "|----------------:";
  if (Test_Mode::pure_hbm == test_mode) {
    //<< "| insert_and_evict "
    cout << "|-----------------:";
  }
  cout << "|\n";
}

void test_main(const size_t dim,
               const size_t init_capacity = 64 * 1024 * 1024UL,
               const size_t key_num_per_op = 1 * 1024 * 1024UL,
               const size_t hbm4values = 16, const float load_factor = 1.0f) {
  std::cout << "|" << rep(1) << fixed << setprecision(2) << load_factor << " ";
  std::vector<API_Select> apis{
      API_Select::insert_or_assign, API_Select::find,
      API_Select::find_or_insert,   API_Select::assign,
      API_Select::find_ptr,         API_Select::find_or_insert_ptr};
  if (Test_Mode::pure_hbm == test_mode) {
    apis.push_back(API_Select::insert_and_evict);
  }
  for (auto api : apis) {
    // There is a sampling of load_factor after several times call to target
    // API. Two consecutive calls can avoid the impact of sampling.
    auto res1 = test_one_api(api, dim, init_capacity, key_num_per_op,
                             hbm4values, load_factor);
    auto res2 = test_one_api(api, dim, init_capacity, key_num_per_op,
                             hbm4values, load_factor);
    auto res = std::max(res1, res2);
    std::cout << "|";
    switch (api) {
      case API_Select::find: {
        std::cout << rep(2);
        break;
      }
      case API_Select::insert_or_assign: {
        std::cout << rep(12);
        break;
      }
      case API_Select::find_or_insert: {
        std::cout << rep(10);
        break;
      }
      case API_Select::assign: {
        std::cout << rep(2);
        break;
      }
      case API_Select::insert_and_evict: {
        std::cout << rep(12);
        break;
      }
      case API_Select::find_ptr: {
        std::cout << rep(2);
        break;
      }
      case API_Select::find_or_insert_ptr: {
        std::cout << rep(11);
        break;
      }
      default: {
        std::cout << "[Unsupport API]";
      }
    }
    std::cout << fixed << setprecision(3) << res << " ";
  }
  std::cout << "|\n";
}

int main() {
  size_t key_num_per_op = 1 * 1024 * 1024UL;
  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));
  cout << endl
       << "## Benchmark" << endl
       << endl
       << "* GPU: 1 x " << props.name << ": " << props.major << "."
       << props.minor << endl
       << "* Key Type = uint64_t" << endl
       << "* Value Type = float32 * {dim}" << endl
       << "* Key-Values per OP = " << key_num_per_op << endl
       << "* Evict strategy: LRU" << endl
       << "* `λ`: load factor" << endl
       << "* `find*` means the `find` API that directly returns the addresses "
          "of values."
       << endl
       << "* `find_or_insert*` means the `find_or_insert` API that directly "
          "returns the addresses of values."
       << endl
       << "* ***Throughput Unit: Billion-KV/second***" << endl
       << endl;
  auto print_configuration = [](const size_t dim, const size_t init_capacity,
                                const size_t hbm4values) {
    using V = float;
    int32_t capacity = static_cast<int32_t>(init_capacity / (1024 * 1024));
    size_t hmem4values = init_capacity * dim * sizeof(V) / (1024 * 1024 * 1024);
    hmem4values = hmem4values < hbm4values ? 0 : (hmem4values - hbm4values);
    cout << "\n* dim = " << dim << ", "
         << "capacity = " << capacity << " Million-KV, "
         << "HBM = " << hbm4values << " GB, "
         << "HMEM = " << hmem4values << " GB\n";
  };
  try {
    test_mode = Test_Mode::pure_hbm;
    cout << "### On pure HBM mode: " << endl;
    print_configuration(4, 64 * 1024 * 1024UL, 32);
    print_title();
    test_main(4, 64 * 1024 * 1024UL, key_num_per_op, 32, 0.50f);
    test_main(4, 64 * 1024 * 1024UL, key_num_per_op, 32, 0.75f);
    test_main(4, 64 * 1024 * 1024UL, key_num_per_op, 32, 1.00f);

    print_configuration(64, 64 * 1024 * 1024UL, 16);
    print_title();
    test_main(64, 64 * 1024 * 1024UL, key_num_per_op, 16, 0.50f);
    test_main(64, 64 * 1024 * 1024UL, key_num_per_op, 16, 0.75f);
    test_main(64, 64 * 1024 * 1024UL, key_num_per_op, 16, 1.00f);
    cout << endl;

    cout << "### On HBM+HMEM hybrid mode: " << endl;
    test_mode = Test_Mode::hybrid;
    print_configuration(64, 128 * 1024 * 1024UL, 16);
    print_title();
    test_main(64, 128 * 1024 * 1024UL, key_num_per_op, 16, 0.50f);
    test_main(64, 128 * 1024 * 1024UL, key_num_per_op, 16, 0.75f);
    test_main(64, 128 * 1024 * 1024UL, key_num_per_op, 16, 1.00f);

    print_configuration(64, 1024 * 1024 * 1024UL, 56);
    print_title();
    test_main(64, 1024 * 1024 * 1024UL, key_num_per_op, 56, 0.50f);
    test_main(64, 1024 * 1024 * 1024UL, key_num_per_op, 56, 0.75f);
    test_main(64, 1024 * 1024 * 1024UL, key_num_per_op, 56, 1.00f);
    cout << endl;

    CUDA_CHECK(hipDeviceSynchronize());
  } catch (const nv::merlin::CudaException& e) {
    cerr << e.what() << endl;
  }
  CUDA_CHECK(hipDeviceSynchronize());
  return 0;
}
