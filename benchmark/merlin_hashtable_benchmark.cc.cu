/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include <thread>
#include <unordered_map>
#include <unordered_set>
#include "merlin/initializers.cuh"
#include "merlin/optimizers.cuh"
#include "merlin_hashtable.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::fixed;
using std::setfill;
using std::setprecision;
using std::setw;

uint64_t getTimestamp() {
  return std::chrono::duration_cast<std::chrono::milliseconds>(
             std::chrono::system_clock::now().time_since_epoch())
      .count();
}
template <class K, class M>
void create_random_keys(K* h_keys, M* h_metas, int key_num_per_op) {
  std::unordered_set<K> numbers;
  std::random_device rd;
  std::mt19937_64 eng(rd());
  std::uniform_int_distribution<K> distr;
  int i = 0;

  while (numbers.size() < key_num_per_op) {
    numbers.insert(distr(eng));
  }
  for (const K num : numbers) {
    h_keys[i] = num;
    h_metas[i] = getTimestamp();
    i++;
  }
}

std::string rep(int n) { return std::string(n, ' '); }

template <class K, class M>
void create_continuous_keys(K* h_keys, M* h_metas, int key_num_per_op,
                            K start = 0) {
  for (K i = 0; i < key_num_per_op; i++) {
    h_keys[i] = start + static_cast<K>(i);
    h_metas[i] = getTimestamp();
  }
}

template <class V, size_t DIM>
struct ValueArray {
  V value[DIM];
};

template <size_t DIM>
void test_main(size_t init_capacity = 64 * 1024 * 1024UL,
               size_t key_num_per_op = 1 * 1024 * 1024UL,
               size_t hbm4values = 16, float load_factor = 1.0) {
  using K = uint64_t;
  using M = uint64_t;
  using Vector = ValueArray<float, DIM>;
  using Table = nv::merlin::HashTable<K, float, M, DIM>;
  using TableOptions = nv::merlin::HashTableOptions;

  size_t free, total;
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMemGetInfo(&free, &total));

  if (free / (1 << 30) < hbm4values) {
    return;
  }

  K* h_keys;
  M* h_metas;
  Vector* h_vectors;
  bool* h_found;

  TableOptions options;

  options.init_capacity = init_capacity;
  options.max_capacity = init_capacity;
  options.max_hbm_for_vectors = nv::merlin::GB(hbm4values);

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(options);

  CUDA_CHECK(hipHostMalloc(&h_keys, key_num_per_op * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_metas, key_num_per_op * sizeof(M)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, key_num_per_op * sizeof(Vector)));
  CUDA_CHECK(hipHostMalloc(&h_found, key_num_per_op * sizeof(bool)));

  CUDA_CHECK(hipMemset(h_vectors, 0, key_num_per_op * sizeof(Vector)));

  K* d_keys;
  M* d_metas = nullptr;
  Vector* d_vectors;
  Vector* d_def_val;
  Vector** d_vectors_ptr;
  bool* d_found;

  CUDA_CHECK(hipMalloc(&d_keys, key_num_per_op * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_metas, key_num_per_op * sizeof(M)));
  CUDA_CHECK(hipMalloc(&d_vectors, key_num_per_op * sizeof(Vector)));
  CUDA_CHECK(hipMalloc(&d_def_val, key_num_per_op * sizeof(Vector)));
  CUDA_CHECK(hipMalloc(&d_vectors_ptr, key_num_per_op * sizeof(Vector*)));
  CUDA_CHECK(hipMalloc(&d_found, key_num_per_op * sizeof(bool)));

  CUDA_CHECK(hipMemcpy(d_keys, h_keys, key_num_per_op * sizeof(K),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_metas, h_metas, key_num_per_op * sizeof(M),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMemset(d_vectors, 1, key_num_per_op * sizeof(Vector)));
  CUDA_CHECK(hipMemset(d_def_val, 2, key_num_per_op * sizeof(Vector)));
  CUDA_CHECK(hipMemset(d_vectors_ptr, 0, key_num_per_op * sizeof(Vector*)));
  CUDA_CHECK(hipMemset(d_found, 0, key_num_per_op * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  K start = 0UL;
  float cur_load_factor = table->load_factor();
  auto start_insert_or_assign = std::chrono::steady_clock::now();
  auto end_insert_or_assign = std::chrono::steady_clock::now();
  auto start_find = std::chrono::steady_clock::now();
  auto end_find = std::chrono::steady_clock::now();
  std::chrono::duration<double> diff_insert_or_assign;
  std::chrono::duration<double> diff_find;

  while (cur_load_factor < load_factor) {
    create_continuous_keys<K, M>(h_keys, h_metas, key_num_per_op, start);
    CUDA_CHECK(hipMemcpy(d_keys, h_keys, key_num_per_op * sizeof(K),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_metas, h_metas, key_num_per_op * sizeof(M),
                          hipMemcpyHostToDevice));

    start_insert_or_assign = std::chrono::steady_clock::now();
    table->insert_or_assign(key_num_per_op, d_keys,
                            reinterpret_cast<float*>(d_vectors), d_metas,
                            stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    end_insert_or_assign = std::chrono::steady_clock::now();
    diff_insert_or_assign = end_insert_or_assign - start_insert_or_assign;

    start_find = std::chrono::steady_clock::now();
    table->find(key_num_per_op, d_keys, reinterpret_cast<float*>(d_vectors),
                d_found, nullptr, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    end_find = std::chrono::steady_clock::now();
    diff_find = end_find - start_find;

    cur_load_factor = table->load_factor(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    start += key_num_per_op;
  }

  size_t hmem4values =
      init_capacity * DIM * sizeof(float) / (1024 * 1024 * 1024);
  hmem4values = hmem4values < hbm4values ? 0 : (hmem4values - hbm4values);
  float insert_tput =
      key_num_per_op / diff_insert_or_assign.count() / (1024 * 1024 * 1024.0);
  float find_tput = key_num_per_op / diff_find.count() / (1024 * 1024 * 1024.0);

  cout << "|" << rep(1) << setw(3) << setfill(' ') << DIM << " "
       << "|" << rep(1) << setw(11) << setfill(' ') << init_capacity << " "
       << "|" << rep(8) << fixed << setprecision(2) << load_factor << " "
       << "|" << rep(5) << setw(3) << setfill(' ') << hbm4values << " "
       << "|" << rep(6) << setw(3) << setfill(' ') << hmem4values << " "
       << "|" << rep(2) << fixed << setprecision(3) << insert_tput << " "
       << "|" << rep(2) << fixed << setprecision(3) << find_tput << " |"
       << endl;

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_metas));
  CUDA_CHECK(hipHostFree(h_found));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_metas));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_def_val));
  CUDA_CHECK(hipFree(d_vectors_ptr));
  CUDA_CHECK(hipFree(d_found));

  CudaCheckError();

  return;
}

void print_title() {
  cout << endl
       << "| dim "
       << "|    capacity "
       << "| load_factor "
       << "| HBM(GB) "
       << "| HMEM(GB) "
       << "| insert "
       << "|   find |" << endl;
  cout << "|----:"
       //<< "| capacity "
       << "|------------:"
       //<< "| load_factor "
       << "|------------:"
       //<< "| HBM(GB) "
       << "|--------:"
       //<< "| HMEM(GB) "
       << "|---------:"
       //<< "| insert "
       << "|-------:"
       //<< "|  find "
       << "|-------:|" << endl;
}

int main() {
  size_t key_num_per_op = 1 * 1024 * 1024UL;
  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));
  cout << endl
       << "## Benchmark" << endl
       << endl
       << "* GPU: 1 x " << props.name << ": " << props.major << "."
       << props.minor << endl
       << "* Key Type = uint64_t" << endl
       << "* Value Type = float32 * dim" << endl
       << "* Key-Values per OP = " << key_num_per_op << endl
       << "* ***Throughput Unit: Billion-KV/second***" << endl
       << endl
       << "### On pure HBM mode: " << endl;
  print_title();
  try {
    test_main<4>(64 * 1024 * 1024UL, key_num_per_op, 32, 0.50);
    test_main<4>(64 * 1024 * 1024UL, key_num_per_op, 32, 0.75);
    test_main<4>(64 * 1024 * 1024UL, key_num_per_op, 32, 1.00);

    test_main<16>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.50);
    test_main<16>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.75);
    test_main<16>(64 * 1024 * 1024UL, key_num_per_op, 16, 1.00);

    test_main<64>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.50);
    test_main<64>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.75);
    test_main<64>(64 * 1024 * 1024UL, key_num_per_op, 16, 1.00);

    test_main<128>(128 * 1024 * 1024UL, key_num_per_op, 64, 0.50);
    test_main<128>(128 * 1024 * 1024UL, key_num_per_op, 64, 0.75);
    test_main<128>(128 * 1024 * 1024UL, key_num_per_op, 64, 1.00);
    cout << endl;

    cout << "### On HBM+HMEM hybrid mode: " << endl;
    print_title();
    test_main<64>(128 * 1024 * 1024UL, key_num_per_op, 16, 0.50);
    test_main<64>(128 * 1024 * 1024UL, key_num_per_op, 16, 0.75);
    test_main<64>(128 * 1024 * 1024UL, key_num_per_op, 16, 1.00);

    test_main<64>(1024 * 1024 * 1024UL, key_num_per_op, 56, 0.50);
    test_main<64>(1024 * 1024 * 1024UL, key_num_per_op, 56, 0.75);
    test_main<64>(1024 * 1024 * 1024UL, key_num_per_op, 56, 1.00);

    test_main<128>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.50);
    test_main<128>(64 * 1024 * 1024UL, key_num_per_op, 16, 0.75);
    test_main<128>(64 * 1024 * 1024UL, key_num_per_op, 16, 1.00);

    test_main<128>(512 * 1024 * 1024UL, key_num_per_op, 56, 0.50);
    test_main<128>(512 * 1024 * 1024UL, key_num_per_op, 56, 0.75);
    test_main<128>(512 * 1024 * 1024UL, key_num_per_op, 56, 1.00);
    cout << endl;

    CUDA_CHECK(hipDeviceSynchronize());
  } catch (const nv::merlin::CudaException& e) {
    cerr << e.what() << endl;
  }
  CUDA_CHECK(hipDeviceSynchronize());
  return 0;
}
