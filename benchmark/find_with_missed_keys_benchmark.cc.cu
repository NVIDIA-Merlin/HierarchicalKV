/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>

#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <limits>
#include <random>
#include <string>
#include <thread>
#include <unordered_map>
#include <unordered_set>

#include "benchmark_util.cuh"
#include "merlin_hashtable.cuh"

using K = uint64_t;
using V = float;
using S = uint64_t;
using EvictStrategy = nv::merlin::EvictStrategy;
using TableOptions = nv::merlin::HashTableOptions;
using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

void print_tile() {
  std::cout << std::endl
            << "|    \u03BB "
            << "| capacity "
            << "| max_hbm_for_vectors "
            << "| max_bucket_size "
            << "| dim "
            << "| missed_ratio "
            << "| througput(BillionKV/secs) ";
  std::cout << "|\n";

  //<< "| load_factor "
  std::cout << "|------"
            //<< "| capacity "
            << "|----------"
            //<< "| max_hbm_for_vectors "
            << "|---------------------"
            //<< "| max_bucket_size "
            << "|-----------------"
            //<< "| dim "
            << "|-----"
            //<< "| missed_ratio "
            << "|--------------"
            //<< "| througput(BillionKV/secs) "
            << "|---------------------------";
  std::cout << "|\n";
}

template <typename T>
void print_w(const T& t, size_t width) {
  std::cout << "|" << std::setw(width) << t;
}

void print_result(double load_factor, size_t capacity,
                  size_t max_hbm_for_vectors, size_t max_bucket_size,
                  size_t dim, double missed_ratio, float througput) {
  print_w(load_factor, 6);
  print_w(capacity, 10);
  print_w(max_hbm_for_vectors, 21);
  print_w(max_bucket_size, 17);
  print_w(dim, 5);
  print_w(missed_ratio, 14);
  print_w(througput, 27);
  std::cout << "|\n";
}

void test_find(size_t capacity, size_t dim, size_t max_hbm_for_vectors,
               double load_factor, size_t max_bucket_size,
               double missed_ratio) {
  MERLIN_CHECK(load_factor >= 0.0 && load_factor <= 1.0,
               "Invalid `load_factor`");
  K* h_keys;
  S* h_scores;
  V* h_vectors;

  TableOptions options;
  options.init_capacity = capacity;
  options.max_capacity = capacity;
  options.dim = dim;

  options.max_hbm_for_vectors = nv::merlin::MB(max_hbm_for_vectors);
  options.max_bucket_size = max_bucket_size;

  size_t key_num = capacity;
  CUDA_CHECK(hipHostMalloc(&h_keys, key_num * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, key_num * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, key_num * options.dim * sizeof(V)));

  K* d_keys;
  S* d_scores;
  V* d_vectors;
  K* d_missed_keys;
  int* d_missed_indices;
  int* d_missed_size;

  CUDA_CHECK(hipMalloc(&d_keys, key_num * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, key_num * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, key_num * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_missed_keys, key_num * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_missed_indices, key_num * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_missed_size, sizeof(int)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  // insert key-value
  size_t insert_num = (double)key_num * load_factor;
  benchmark::create_continuous_keys<K, S>(h_keys, h_scores, insert_num,
                                          0 /*start*/);
  benchmark::init_value_using_key<K, V>(h_keys, h_vectors, insert_num,
                                        options.dim);
  CUDA_CHECK(hipMemcpy(d_keys, h_keys, insert_num * sizeof(K),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_scores, h_scores, insert_num * sizeof(S),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
                        insert_num * sizeof(V) * options.dim,
                        hipMemcpyHostToDevice));
  Table table;
  table.init(options);
  table.insert_or_assign(insert_num, d_keys, d_vectors, d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  // find key-value
  size_t find_num = (double)insert_num * (1.0 - missed_ratio);
  benchmark::create_continuous_keys<K, S>(h_keys, nullptr, find_num,
                                          0 /*start*/);
  benchmark::create_continuous_keys<K, S>(
      h_keys + find_num, nullptr, insert_num - find_num, insert_num /*start*/);
  CUDA_CHECK(hipMemcpy(d_keys, h_keys, insert_num * sizeof(K),
                        hipMemcpyHostToDevice));

  auto timer = benchmark::Timer<double>();
  timer.start();
  table.find(insert_num, d_keys, d_vectors, d_missed_keys, d_missed_indices,
             d_missed_size, d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  timer.end();

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_vectors));
  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_missed_keys));
  CUDA_CHECK(hipFree(d_missed_indices));
  CUDA_CHECK(hipFree(d_missed_size));

  CudaCheckError();
  float througput = insert_num / timer.getResult() / (1024 * 1024 * 1024.0f);
  print_result(load_factor, capacity, max_hbm_for_vectors, max_bucket_size, dim,
               missed_ratio, througput);
}

void test_main(double load_factor, double missed_ratio) {
  constexpr size_t CAPACITY = 100000000UL;
  print_tile();
  // pure HBM
  test_find(CAPACITY, 8, 8 * 1024UL, load_factor, 256, missed_ratio);
  test_find(CAPACITY, 8, 8 * 1024UL, load_factor, 128, missed_ratio);
  // hybrid
  test_find(CAPACITY, 8, 1 * 1024UL, load_factor, 256, missed_ratio);
  test_find(CAPACITY, 8, 1 * 1024UL, load_factor, 128, missed_ratio);
  // pure HMEM
  test_find(CAPACITY, 8, 0, load_factor, 256, missed_ratio);
  test_find(CAPACITY, 8, 0, load_factor, 128, missed_ratio);
}

int main() {
  test_main(0.2, 0);
  test_main(0.2, 0.5);
  test_main(0.2, 1.0);
  test_main(0.5, 0);
  test_main(0.5, 0.5);
  test_main(0.5, 1.0);
  test_main(1.0, 0);
  test_main(1.0, 0.5);
  test_main(1.0, 1.0);
  return 0;
}
