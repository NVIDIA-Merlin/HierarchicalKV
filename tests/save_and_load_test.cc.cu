/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <stdio.h>
#include "merlin/types.cuh"
#include "merlin_hashtable.cuh"
#include "merlin_localfile.hpp"
#include "test_util.cuh"

constexpr uint64_t DIM = 64;
using K = int64_t;
using S = uint64_t;
using V = float;
using Table = nv::merlin::HashTable<K, V, S>;
using TableOptions = nv::merlin::HashTableOptions;

void test_save_to_file() {
  std::string prefix = "checkpoint";
  size_t keynum = 1 * 1024 * 1024;
  size_t capacity = 2 * 1024 * 1024;
  size_t buffer_size = 1024 * 1024;
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  K* h_keys = nullptr;
  V* h_vectors = nullptr;
  S* h_scores = nullptr;
  CUDA_CHECK(hipHostMalloc(&h_keys, keynum * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, keynum * sizeof(V) * DIM));
  CUDA_CHECK(hipHostMalloc(&h_scores, keynum * sizeof(S)));
  memset(h_keys, 0, keynum * sizeof(K));
  memset(h_vectors, 0, keynum * sizeof(V) * DIM);
  memset(h_scores, 0, keynum * sizeof(S));
  test_util::create_random_keys<K, S>(h_keys, h_scores, keynum);
  printf("Pass create random keys.\n");

  K* d_keys = nullptr;
  V* d_vectors = nullptr;
  S* d_scores = nullptr;
  test_util::getBufferOnDevice(&d_keys, keynum * sizeof(K), stream);
  test_util::getBufferOnDevice(&d_vectors, keynum * sizeof(V) * DIM, stream);
  test_util::getBufferOnDevice(&d_scores, keynum * sizeof(S), stream);
  CUDA_CHECK(hipMemcpyAsync(d_keys, h_keys, keynum * sizeof(K),
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(d_vectors, h_vectors, keynum * sizeof(V) * DIM,
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(d_scores, h_scores, keynum * sizeof(S),
                             hipMemcpyHostToDevice, stream));
  printf("Create buffers.\n");

  TableOptions options;
  options.init_capacity = capacity;
  options.max_capacity = capacity;
  options.dim = DIM;
  options.evict_strategy = nv::merlin::EvictStrategy::kLru;

  std::unique_ptr<Table> table_0 = std::make_unique<Table>();
  std::unique_ptr<Table> table_1 = std::make_unique<Table>();
  table_0->init(options);
  table_1->init(options);
  printf("Init tables.\n");

  table_0->insert_or_assign(keynum, d_keys, d_vectors, /*scores=*/nullptr,
                            stream);
  printf("Fill table_0.\n");
  nv::merlin::LocalKVFile<K, V, S> file;
  std::string keys_path = prefix + ".keys";
  std::string values_path = prefix + ".values";
  std::string scores_path = prefix + ".scores";
  file.open(keys_path, values_path, scores_path, "wb");
  table_0->save(&file, buffer_size, stream);
  file.close();
  printf("table_0 saves.\n");
  file.open(keys_path, values_path, scores_path, "rb");
  table_1->load(&file, buffer_size, stream);
  file.close();
  printf("table_1 loads.\n");
  ASSERT_TRUE(test_util::tables_equal(table_0.get(), table_1.get(), stream));
  printf("table_0 and table_1 are equal.\n");
  CUDA_FREE_POINTERS(stream, d_keys, d_vectors, d_scores, h_keys, h_vectors,
                     h_scores);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

TEST(SaveAndLoadTest, test_save_and_load) { test_save_to_file(); }
