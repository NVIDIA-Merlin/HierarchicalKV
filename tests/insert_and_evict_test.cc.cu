/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <stdio.h>
#include <array>
#include <map>
#include "merlin/types.cuh"
#include "merlin_hashtable.cuh"
#include "merlin_localfile.hpp"
#include "test_util.cuh"

constexpr size_t dim = 64;
using i64 = int64_t;
using u64 = uint64_t;
using f32 = float;
using Table = nv::merlin::HashTable<i64, f32, u64>;
using TableOptions = nv::merlin::HashTableOptions;

/*
 * There are several steps to check whether if
 * the insert_and_evict API is safe to use:
 *
 *   step1: Create a table with max_capacity U
 *   step2: Insert M keys into table while M < U. And
 *     the table size became m <= M. M - m keys was
 *     evicted.
 *   step3: Insert N keys into table while m + N > U, with
 *     no same key with M keys. And p keys gets evicted.
 *     If now the table size is v. Then total number of
 *     keys T = v + p + M - m, must equal to VT = M + N,
 *     while the keys, values, and metas match.
 *   step4: export table and check all values.
 */
void test_insert_and_evict_basic() {
  TableOptions opt;

  // table setting
  const size_t init_capacity = 1024;

  // numeric setting
  const size_t U = 2llu << 18;
  const size_t M = (U >> 1);
  const size_t N = (U >> 1) + 17;  // Add a prime to test the non-aligned case.

  opt.max_capacity = U;
  opt.init_capacity = init_capacity;
  opt.max_hbm_for_vectors = U * dim * sizeof(f32);
  opt.evict_strategy = nv::merlin::EvictStrategy::kCustomized;
  opt.dim = dim;

  std::map<i64, test_util::ValueArray<f32, dim>> summarized_kvs;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // step1
  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(opt);

  // step2
  test_util::KVMSBuffer<i64, f32, u64> evict_buffer;
  evict_buffer.Reserve(M, dim, stream);
  evict_buffer.ToZeros(stream);

  test_util::KVMSBuffer<i64, f32, u64> buffer;
  buffer.Reserve(M, dim, stream);
  buffer.ToRange(0, 1, stream);
  buffer.SetMeta((u64)1, stream);
  size_t n_evicted = table->insert_and_evict(
      M, buffer.keys_ptr(), buffer.values_ptr(), buffer.metas_ptr(),
      evict_buffer.keys_ptr(), evict_buffer.values_ptr(),
      evict_buffer.metas_ptr(), stream);
  size_t table_size_m = table->size(stream);
  buffer.SyncData(/*h2d=*/false, stream);
  evict_buffer.SyncData(/*h2d=*/false, stream);
  ASSERT_EQ(n_evicted + table_size_m, M);
  for (size_t i = 0; i < n_evicted; i++) {
    test_util::ValueArray<f32, dim>* vec =
        reinterpret_cast<test_util::ValueArray<f32, dim>*>(
            evict_buffer.values_ptr(false) + i * dim);
    summarized_kvs.emplace(evict_buffer.keys_ptr(false)[i], *vec);
  }
  CUDA_CHECK(hipStreamSynchronize(stream));

  //  step3
  evict_buffer.Reserve(N, dim, stream);
  buffer.Reserve(N, dim, stream);
  buffer.ToRange(M, 1, stream);
  buffer.SetMeta((u64)2, stream);
  n_evicted = table->insert_and_evict(
      N, buffer.keys_ptr(), buffer.values_ptr(), buffer.metas_ptr(),
      evict_buffer.keys_ptr(), evict_buffer.values_ptr(),
      evict_buffer.metas_ptr(), stream);
  size_t table_size_n = table->size(stream);
  buffer.SyncData(/*h2d=*/false, stream);
  evict_buffer.SyncData(/*h2d=*/false, stream);
  ASSERT_EQ(table_size_m + N, table_size_n + n_evicted);
  for (size_t i = 0; i < n_evicted; i++) {
    test_util::ValueArray<f32, dim>* vec =
        reinterpret_cast<test_util::ValueArray<f32, dim>*>(
            evict_buffer.values_ptr(false) + i * dim);
    summarized_kvs.emplace(evict_buffer.keys_ptr(false)[i], *vec);
  }
  CUDA_CHECK(hipStreamSynchronize(stream));

  // step4
  buffer.Reserve(table_size_n, dim, stream);
  size_t n_exported =
      table->export_batch(table->capacity(), 0, buffer.keys_ptr(),
                          buffer.values_ptr(), buffer.metas_ptr(), stream);
  ASSERT_EQ(table_size_n, n_exported);
  buffer.SyncData(/*h2d=*/false, stream);
  for (size_t i = 0; i < n_exported; i++) {
    test_util::ValueArray<f32, dim>* vec =
        reinterpret_cast<test_util::ValueArray<f32, dim>*>(
            buffer.values_ptr(false) + i * dim);
    summarized_kvs.emplace(buffer.keys_ptr(false)[i], *vec);
  }

  CUDA_CHECK(hipStreamSynchronize(stream));
  buffer.Free(stream);
  evict_buffer.Free(stream);

  size_t k = 0;
  for (auto it = summarized_kvs.begin(); it != summarized_kvs.end(); it++) {
    i64 key = it->first;
    test_util::ValueArray<f32, dim>& value = it->second;
    ASSERT_EQ(key, (i64)k);
    for (size_t j = 0; j < dim; j++) {
      ASSERT_EQ(value[j], (f32)k);
    }
    ++k;
  }
  ASSERT_EQ(summarized_kvs.size(), M + N);
  summarized_kvs.clear();
}

template <typename K, typename V, typename M>
void CheckInsertAndEvict(Table* table, K* keys, V* values, M* metas,
                         K* evicted_keys, V* evicted_values, M* evicted_metas,
                         size_t len, hipStream_t stream) {
  std::map<i64, test_util::ValueArray<f32, dim>> map_before_insert;
  std::map<i64, test_util::ValueArray<f32, dim>> map_after_insert;
  K* h_tmp_keys = nullptr;
  V* h_tmp_values = nullptr;
  M* h_tmp_metas = nullptr;

  K* d_tmp_keys = nullptr;
  V* d_tmp_values = nullptr;
  M* d_tmp_metas = nullptr;

  size_t table_size_before = table->size(stream);
  size_t cap = table_size_before + len;

  CUDA_CHECK(hipMallocAsync(&d_tmp_keys, cap * sizeof(K), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_keys, 0, cap * sizeof(K), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_values, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_values, 0, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_metas, cap * sizeof(M), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_metas, 0, cap * sizeof(M), stream));
  h_tmp_keys = (K*)malloc(cap * sizeof(K));
  h_tmp_values = (V*)malloc(cap * dim * sizeof(V));
  h_tmp_metas = (M*)malloc(cap * sizeof(M));

  size_t table_size_verify0 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_metas, stream);
  ASSERT_EQ(table_size_before, table_size_verify0);

  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_before * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_before * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_metas, d_tmp_metas,
                             table_size_before * sizeof(M),
                             hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys + table_size_before, keys,
                             len * sizeof(K), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values + table_size_before * dim, values,
                             len * dim * sizeof(V), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_metas + table_size_before, metas,
                             len * sizeof(M), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  for (size_t i = 0; i < cap; i++) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    map_before_insert[h_tmp_keys[i]] = *vec;
  }

  auto start = std::chrono::steady_clock::now();
  size_t filtered_len =
      table->insert_and_evict(len, keys, values, nullptr, evicted_keys,
                              evicted_values, evicted_metas, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  auto end = std::chrono::steady_clock::now();
  auto diff = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

  float dur = diff.count();

  size_t table_size_after = table->size(stream);
  size_t table_size_verify1 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_metas, stream);

  ASSERT_EQ(table_size_verify1, table_size_after);

  size_t new_cap = table_size_after + filtered_len;
  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_after * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_after * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_metas, d_tmp_metas,
                             table_size_after * sizeof(M),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys + table_size_after, evicted_keys,
                             filtered_len * sizeof(K), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values + table_size_after * dim,
                             evicted_values, filtered_len * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_metas + table_size_after, evicted_metas,
                             filtered_len * sizeof(M), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipStreamSynchronize(stream));
  int64_t new_cap_i64 = (int64_t)new_cap;
  for (int64_t i = new_cap_i64 - 1; i >= 0; i--) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    map_after_insert[h_tmp_keys[i]] = *vec;
  }

  size_t key_miss_cnt = 0;
  size_t value_diff_cnt = 0;
  for (auto& it : map_before_insert) {
    if (map_after_insert.find(it.first) == map_after_insert.end()) {
      ++key_miss_cnt;
      continue;
    }
    test_util::ValueArray<V, dim>& vec0 = it.second;
    test_util::ValueArray<V, dim>& vec1 = map_after_insert.at(it.first);
    for (size_t j = 0; j < dim; j++) {
      if (vec0[j] != vec1[j]) {
        ++value_diff_cnt;
        break;
      }
    }
  }
  ASSERT_EQ(key_miss_cnt, 0);
  ASSERT_EQ(value_diff_cnt, 0);

  CUDA_CHECK(hipFreeAsync(d_tmp_keys, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_values, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_metas, stream));
  free(h_tmp_keys);
  free(h_tmp_values);
  free(h_tmp_metas);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

void test_insert_and_evict_advanced() {
  const size_t U = 524288;
  const size_t init_capacity = 1024;
  const size_t B = 524288 + 13;

  TableOptions opt;

  opt.max_capacity = U;
  opt.init_capacity = init_capacity;
  opt.max_hbm_for_vectors = U * dim * sizeof(f32);
  opt.evict_strategy = nv::merlin::EvictStrategy::kLru;
  opt.dim = dim;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(opt);

  test_util::KVMSBuffer<i64, f32, u64> evict_buffer;
  evict_buffer.Reserve(B, dim, stream);
  evict_buffer.ToZeros(stream);

  test_util::KVMSBuffer<i64, f32, u64> data_buffer;
  data_buffer.Reserve(B, dim, stream);

  size_t offset = 0;
  u64 meta = 0;
  for (int i = 0; i < 20; i++) {
    test_util::create_random_keys<i64, u64, f32, dim>(
        data_buffer.keys_ptr(false), data_buffer.metas_ptr(false),
        data_buffer.values_ptr(false), (int)B, B * 16);
    data_buffer.SyncData(true, stream);

    CheckInsertAndEvict<i64, f32, u64>(
        table.get(), data_buffer.keys_ptr(), data_buffer.values_ptr(),
        data_buffer.metas_ptr(), evict_buffer.keys_ptr(),
        evict_buffer.values_ptr(), evict_buffer.metas_ptr(), B, stream);

    offset += B;
    meta += 1;
  }
}

TEST(MerlinHashTableTest, test_insert_and_evict_basic) {
  test_insert_and_evict_basic();
}
TEST(MerlinHashTableTest, test_insert_and_evict_advanced) {
  test_insert_and_evict_advanced();
}
