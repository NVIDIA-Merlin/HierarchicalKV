#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <array>
#include <iostream>
#include <thread>
#include <unordered_map>
#include <vector>
#include "merlin_hashtable.cuh"
#include "test_util.cuh"

constexpr size_t DIM = 16;
using K = uint64_t;
using V = float;
using S = uint64_t;
using TableOptions = nv::merlin::HashTableOptions;
using EvictStrategy = nv::merlin::EvictStrategy;

template <class K, class S>
struct EraseIfPredFunctor {
  __forceinline__ __device__ bool operator()(const K& key, S& score,
                                             const K& pattern,
                                             const S& threshold) {
    return (((key & 0x1u) == 0x1u) && (score > threshold));
  }
};

template <class K, class S>
struct ExportIfPredFunctor {
  __forceinline__ __device__ bool operator()(const K& key, S& score,
                                             const K& pattern,
                                             const S& threshold) {
    return score > threshold;
  }
};

void test_basic_when_full(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t INIT_CAPACITY = 1 * 1024 * 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t KEY_NUM = 1 * 1024 * 1024UL;
  constexpr uint64_t TEST_TIMES = 1;

  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_accum_or_assigns;
  bool* h_found;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));

  CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));

  test_util::create_random_keys<K, S, V, DIM>(h_keys, h_scores, nullptr,
                                              KEY_NUM);

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  bool* d_accum_or_assigns;
  V* d_def_val;
  V** d_vectors_ptr;
  bool* d_found;

  CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_def_val, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_vectors_ptr, KEY_NUM * sizeof(V*)));
  CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));

  CUDA_CHECK(
      hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_scores, h_scores, KEY_NUM * sizeof(S),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMemset(d_vectors, 1, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMemset(d_def_val, 2, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMemset(d_vectors_ptr, 0, KEY_NUM * sizeof(V*)));
  CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  uint64_t total_size = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);
    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                           d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    uint64_t total_size_after_insert = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    table->erase(KEY_NUM, d_keys, stream);
    size_t total_size_after_erase = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size_after_erase, 0);

    table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                           d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    uint64_t total_size_after_reinsert = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size_after_insert, total_size_after_reinsert);
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_found));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_accum_or_assigns));
  CUDA_CHECK(hipFree(d_def_val));
  CUDA_CHECK(hipFree(d_vectors_ptr));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_erase_if_pred(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t INIT_CAPACITY = 256UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t KEY_NUM = 128UL;
  constexpr uint64_t TEST_TIMES = 1;
  constexpr float true_ratio = 0.5;

  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_found;
  bool* h_accum_or_assigns;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(options);

  CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  bool* d_found;
  bool* d_accum_or_assigns;

  CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  uint64_t total_size = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    test_util::create_keys_in_one_buckets<K, S, V, DIM>(
        h_keys, h_scores, h_vectors, KEY_NUM, INIT_CAPACITY);

    test_util::create_random_bools<K>(h_accum_or_assigns, KEY_NUM, true_ratio);
    CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_scores, h_scores, KEY_NUM * sizeof(S),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                          KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                           d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    size_t expected_size = 0;
    for (size_t i = 0; i < KEY_NUM; i++) {
      if (!h_accum_or_assigns[i]) expected_size++;
    }
    ASSERT_EQ(total_size, expected_size);

    K pattern = 100;
    S threshold = 0;
    size_t erase_num = table->template erase_if<EraseIfPredFunctor>(
        pattern, threshold, stream);
    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ((erase_num + total_size), expected_size);

    CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
    table->find(KEY_NUM, d_keys, d_vectors, d_found, d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    int found_num = 0;

    CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipMemset(h_scores, 0, KEY_NUM * sizeof(S)));
    CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
    CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_scores, d_scores, KEY_NUM * sizeof(S),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyDeviceToHost));
    for (int i = 0; i < KEY_NUM; i++) {
      if (h_found[i]) {
        found_num++;
        ASSERT_EQ(h_scores[i], h_keys[i]);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors[i * options.dim + j],
                    static_cast<float>(h_keys[i] * 0.00001));
        }
      }
    }
    ASSERT_EQ(found_num, (expected_size - erase_num));

    table->clear(stream);
    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_found));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipFree(d_accum_or_assigns));
  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_rehash(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t BUCKET_MAX_SIZE = 128ul;
  constexpr uint64_t INIT_CAPACITY = BUCKET_MAX_SIZE;
  constexpr uint64_t MAX_CAPACITY = 4 * INIT_CAPACITY;
  constexpr uint64_t KEY_NUM = BUCKET_MAX_SIZE * 2;
  constexpr uint64_t TEST_TIMES = 100;
  constexpr float true_ratio = 0.5;

  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_found;
  bool* h_accum_or_assigns;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_bucket_size = BUCKET_MAX_SIZE;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  bool* d_found;
  bool* d_accum_or_assigns;
  size_t dump_counter = 0;

  CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  uint64_t total_size = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    test_util::create_random_bools<K>(h_accum_or_assigns, KEY_NUM, true_ratio);
    test_util::create_keys_in_one_buckets<K, S, V, DIM>(
        h_keys, h_scores, h_vectors, KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE);
    CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_scores, h_scores, KEY_NUM * sizeof(S),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                          KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                           d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    total_size = table->size(stream);
    CUDA_CHECK(hipDeviceSynchronize());

    size_t expected_size = 0;
    for (size_t i = 0; i < KEY_NUM; i++) {
      if (!h_accum_or_assigns[i]) expected_size++;
    }
    ASSERT_EQ(total_size, expected_size);

    dump_counter = table->export_batch(table->capacity(), 0, d_keys, d_vectors,
                                       d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(dump_counter, expected_size);

    table->reserve(MAX_CAPACITY, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(table->capacity(), MAX_CAPACITY);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, expected_size);

    CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
    table->find(KEY_NUM, d_keys, d_vectors, d_found, d_scores, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    int found_num = 0;

    CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipMemset(h_scores, 0, KEY_NUM * sizeof(S)));
    CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
    CUDA_CHECK(hipMemcpy(h_keys, d_keys, KEY_NUM * sizeof(K),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_scores, d_scores, KEY_NUM * sizeof(S),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyDeviceToHost));
    for (int i = 0; i < KEY_NUM; i++) {
      if (h_found[i]) {
        found_num++;
        ASSERT_EQ(h_scores[i], h_keys[i]);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors[i * options.dim + j],
                    static_cast<float>(h_keys[i] * 0.00001));
        }
      }
    }
    ASSERT_EQ(found_num, expected_size);

    table->clear(stream);
    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_found));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipFree(d_accum_or_assigns));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_rehash_on_big_batch(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t INIT_CAPACITY = 1024;
  constexpr uint64_t MAX_CAPACITY = 16 * 1024;
  constexpr uint64_t INIT_KEY_NUM = 1024;
  constexpr uint64_t KEY_NUM = 2048;

  std::unordered_map<K, float> expected_values;
  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_found;
  bool* h_accum_or_assigns;
  bool* h_accum_or_assigns_init;
  float true_ratio = 0.6f;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_bucket_size = 128;
  options.max_load_factor = 0.6;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns_init, KEY_NUM * sizeof(bool)));

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  bool* d_accum_or_assigns;
  bool* d_found;
  size_t dump_counter = 0;

  CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  uint64_t total_size = 0;
  uint64_t expected_size = 0;
  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(options);

  test_util::create_random_keys<K, S, V, DIM>(h_keys, h_scores, h_vectors,
                                              KEY_NUM);

  CUDA_CHECK(
      hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_scores, h_scores, KEY_NUM * sizeof(S),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_vectors, h_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMemset(d_accum_or_assigns, 0, KEY_NUM * sizeof(bool)));

  total_size = table->size(stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  ASSERT_EQ(total_size, 0);

  test_util::create_random_bools<K>(h_accum_or_assigns, INIT_KEY_NUM,
                                    true_ratio);
  CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                        INIT_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

  table->accum_or_assign(INIT_KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                         d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  ASSERT_GE(table->capacity(), INIT_CAPACITY * 2);

  expected_size = 0;
  for (int i = 0; i < INIT_KEY_NUM; i++)
    expected_size += (h_accum_or_assigns[i] ? 0 : 1);

  total_size = table->size(stream);
  CUDA_CHECK(hipDeviceSynchronize());
  ASSERT_EQ(total_size, expected_size);
  ASSERT_EQ(table->capacity(), (INIT_CAPACITY * 2));

  CUDA_CHECK(hipMemcpy(h_accum_or_assigns_init, h_accum_or_assigns,
                        KEY_NUM * sizeof(bool), hipMemcpyHostToHost));
  test_util::create_random_bools<K>(h_accum_or_assigns, KEY_NUM, true_ratio);
  CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                        KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

  table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                         d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  expected_size = 0;
  for (int i = 0; i < KEY_NUM; i++) {
    if (i < INIT_KEY_NUM) {
      if (h_accum_or_assigns_init[i]) {
        if (h_accum_or_assigns[i]) {
        } else {
          expected_size++;
          expected_values[h_keys[i]] = static_cast<float>(h_keys[i] * 0.00001);
        }
      } else {
        expected_size++;
        if (h_accum_or_assigns[i]) {
          expected_values[h_keys[i]] = static_cast<float>(h_keys[i] * 0.00002);
        } else {
          expected_values[h_keys[i]] = static_cast<float>(h_keys[i] * 0.00001);
        }
      }
    }
    if (i >= INIT_KEY_NUM && (!h_accum_or_assigns[i])) {
      expected_size++;
      expected_values[h_keys[i]] = static_cast<float>(h_keys[i] * 0.00001);
    }
  }

  total_size = table->size(stream);
  CUDA_CHECK(hipDeviceSynchronize());
  ASSERT_EQ(total_size, expected_size);
  ASSERT_GE(table->capacity(), KEY_NUM * 2);

  dump_counter = table->export_batch(table->capacity(), 0, d_keys, d_vectors,
                                     d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  ASSERT_EQ(dump_counter, expected_size);

  CUDA_CHECK(
      hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMemset(d_scores, 0, KEY_NUM * sizeof(S)));
  table->find(KEY_NUM, d_keys, d_vectors, d_found, d_scores, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  int found_num = 0;

  CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMemset(h_scores, 0, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(
      hipMemcpy(h_keys, d_keys, KEY_NUM * sizeof(K), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_scores, d_scores, KEY_NUM * sizeof(S),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyDeviceToHost));
  for (int i = 0; i < KEY_NUM; i++) {
    if (h_found[i]) {
      found_num++;
      ASSERT_EQ(h_scores[i], h_keys[i]);
      for (int j = 0; j < options.dim; j++) {
        ASSERT_EQ(h_vectors[i * options.dim + j], expected_values[h_keys[i]]);
      }
    }
  }
  ASSERT_EQ(found_num, expected_size);

  table->clear(stream);
  total_size = table->size(stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  ASSERT_EQ(total_size, 0);
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(V) * options.dim,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns_init));
  CUDA_CHECK(hipHostFree(h_found));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_accum_or_assigns));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}
//
// void test_dynamic_rehash_on_multi_threads(size_t max_hbm_for_vectors) {
//  constexpr uint64_t BUCKET_MAX_SIZE = 128ul;
//  constexpr uint64_t INIT_CAPACITY = 4 * 1024;
//  constexpr uint64_t MAX_CAPACITY = 16 * 1024 * INIT_CAPACITY;
//  constexpr uint64_t KEY_NUM = 256;
//  constexpr uint64_t THREAD_N = 8;
//
//  std::vector<std::thread> threads;
//
//  TableOptions options;
//
//  options.init_capacity = INIT_CAPACITY;
//  options.max_capacity = MAX_CAPACITY;
//  options.dim = DIM;
//  options.max_load_factor = 0.50f;
//  options.max_bucket_size = BUCKET_MAX_SIZE;
//  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
//  using Table = nv::merlin::HashTable<K, V, S,
//  EvictStrategy::kLru>;
//
//  std::shared_ptr<Table> table = std::make_shared<Table>();
//  table->init(options);
//
//  auto worker_function = [&table, KEY_NUM, options](int task_n) {
//    K* h_keys;
//    V* h_vectors;
//    bool* h_found;
//
//    size_t current_capacity = table->capacity();
//
//    CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
//    CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
//    CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));
//
//    K* d_keys;
//    V* d_vectors;
//    bool* d_found;
//
//    CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
//    CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
//    CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));
//
//    hipStream_t stream;
//    CUDA_CHECK(hipStreamCreate(&stream));
//
//    while (table->capacity() < MAX_CAPACITY) {
//      test_util::create_random_keys<K, S, V, DIM>(h_keys, nullptr, h_vectors,
//                                                  KEY_NUM);
//      CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
//                            hipMemcpyHostToDevice));
//      CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
//                            KEY_NUM * sizeof(V) * options.dim,
//                            hipMemcpyHostToDevice));
//      CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));
//
//      table->accum_or_assign(KEY_NUM, d_keys, d_vectors, nullptr, stream);
//      CUDA_CHECK(hipStreamSynchronize(stream));
//
//      CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
//      table->find(KEY_NUM, d_keys, d_vectors, d_found, nullptr, stream);
//
//      CUDA_CHECK(hipStreamSynchronize(stream));
//      int found_num = 0;
//
//      CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
//      CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
//      CUDA_CHECK(hipMemcpy(h_keys, d_keys, KEY_NUM * sizeof(K),
//                            hipMemcpyDeviceToHost));
//      CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
//                            hipMemcpyDeviceToHost));
//
//      CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
//                            KEY_NUM * sizeof(V) * options.dim,
//                            hipMemcpyDeviceToHost));
//      for (int i = 0; i < KEY_NUM; i++) {
//        if (h_found[i]) {
//          found_num++;
//          for (int j = 0; j < options.dim; j++) {
//            ASSERT_EQ(h_vectors[i * options.dim + j],
//                      static_cast<float>(h_keys[i] * 0.00001));
//          }
//        }
//      }
//      ASSERT_EQ(found_num, KEY_NUM);
//      if (task_n == 0 && current_capacity != table->capacity()) {
//        std::cout << "[test_dynamic_rehash_on_multi_threads] The capacity "
//                     "changed from "
//                  << current_capacity << " to " << table->capacity()
//                  << std::endl;
//        current_capacity = table->capacity();
//      }
//      CUDA_CHECK(hipStreamSynchronize(stream));
//    }
//    CUDA_CHECK(hipStreamDestroy(stream));
//
//    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
//                          KEY_NUM * sizeof(V) * options.dim,
//                          hipMemcpyDeviceToHost));
//
//    CUDA_CHECK(hipHostFree(h_keys));
//    CUDA_CHECK(hipHostFree(h_found));
//    CUDA_CHECK(hipHostFree(h_vectors));
//
//    CUDA_CHECK(hipFree(d_keys));
//    CUDA_CHECK(hipFree(d_vectors));
//    CUDA_CHECK(hipFree(d_found));
//    CUDA_CHECK(hipDeviceSynchronize());
//
//    CudaCheckError();
//  };
//
//  for (int i = 0; i < THREAD_N; ++i)
//    threads.emplace_back(std::thread(worker_function, i));
//
//  for (auto& th : threads) {
//    th.join();
//  }
//  ASSERT_EQ(table->capacity(), MAX_CAPACITY);
//}
//
void test_export_batch_if(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t INIT_CAPACITY = 256UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t KEY_NUM = 128UL;
  constexpr uint64_t TEST_TIMES = 1;
  constexpr float true_ratio = 0.6;

  K* h_keys;
  S* h_scores;
  V* h_vectors;
  bool* h_accum_or_assigns;
  size_t h_dump_counter = 0;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(options);

  CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipHostMalloc(&h_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));

  K* d_keys;
  S* d_scores = nullptr;
  V* d_vectors;
  bool* d_found;
  bool* d_accum_or_assigns;
  size_t* d_dump_counter;
  int found_num = 0;
  bool* h_found;

  CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores, KEY_NUM * sizeof(S)));
  CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_dump_counter, sizeof(size_t)));

  CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  uint64_t total_size = 0;
  K pattern = 100;
  S threshold = test_util::host_nano<S>(stream);

  for (int i = 0; i < TEST_TIMES; i++) {
    test_util::create_random_bools<K>(h_accum_or_assigns, KEY_NUM, true_ratio);
    test_util::create_random_keys<K, S, V, DIM>(h_keys, h_scores, h_vectors,
                                                KEY_NUM);

    CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_scores, h_scores, KEY_NUM * sizeof(S),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                          KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyHostToDevice));

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                           nullptr, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    size_t expected_size = 0;
    for (size_t i = 0; i < KEY_NUM; i++) {
      if (!h_accum_or_assigns[i]) expected_size++;
    }

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, expected_size);

    CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));

    CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
    table->find(KEY_NUM, d_keys, d_vectors, d_found, nullptr, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
                          hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyDeviceToHost));
    found_num = 0;
    for (int i = 0; i < KEY_NUM; i++) {
      if (h_found[i]) {
        found_num++;
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors[i * options.dim + j],
                    static_cast<float>(h_keys[i] * 0.00001));
        }
      }
    }
    ASSERT_EQ(found_num, expected_size);

    table->template export_batch_if<ExportIfPredFunctor>(
        pattern, threshold, table->capacity(), 0, d_dump_counter, d_keys,
        d_vectors, d_scores, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipMemcpy(&h_dump_counter, d_dump_counter, sizeof(size_t),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_scores, d_scores, h_dump_counter * sizeof(S),
                          hipMemcpyDeviceToHost));

    size_t expected_export_count = 0;
    for (int i = 0; i < h_dump_counter; i++) {
      if (h_scores[i] > threshold) expected_export_count++;
    }
    ASSERT_EQ(expected_export_count, h_dump_counter);

    CUDA_CHECK(hipMemcpy(h_keys, d_keys, KEY_NUM * sizeof(K),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_scores, d_scores, KEY_NUM * sizeof(S),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyDeviceToHost));

    for (int i = 0; i < h_dump_counter; i++) {
      ASSERT_GT(h_scores[i], threshold);
      for (int j = 0; j < options.dim; j++) {
        ASSERT_EQ(h_vectors[i * options.dim + j],
                  static_cast<float>(h_keys[i] * 0.00001));
      }
    }

    table->clear(stream);
    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);
  }
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipHostFree(h_keys));
  CUDA_CHECK(hipHostFree(h_scores));
  CUDA_CHECK(hipHostFree(h_found));
  CUDA_CHECK(hipHostFree(h_accum_or_assigns));

  CUDA_CHECK(hipFree(d_keys));
  CUDA_CHECK(hipFree(d_scores));
  CUDA_CHECK(hipFree(d_vectors));
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipFree(d_accum_or_assigns));
  CUDA_CHECK(hipFree(d_dump_counter));
  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_lru_basic(size_t max_hbm_for_vectors) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 128;
  constexpr float true_ratio = 0.5;

  TableOptions options;

  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF);

  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD);

  h_accum_or_assigns_base[72] = false;
  h_accum_or_assigns_base[73] = false;

  h_keys_test[2] = h_keys_base[72];
  h_keys_test[3] = h_keys_base[73];

  h_accum_or_assigns_test[2] = true;
  h_accum_or_assigns_test[3] = false;

  for (int i = 0; i < options.dim; i++) {
    h_vectors_test[2 * options.dim + i] = h_vectors_base[72 * options.dim + i];
    h_vectors_test[3 * options.dim + i] = h_vectors_base[73 * options.dim + i];
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      S start_ts = test_util::host_nano<S>(stream);
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      S end_ts = test_util::host_nano<S>(stream);

      size_t total_size = table->size(stream);
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        ASSERT_GE(h_scores_temp[i], start_ts);
        ASSERT_LE(h_scores_temp[i], end_ts);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      S start_ts = test_util::host_nano<S>(stream);
      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      S end_ts = test_util::host_nano<S>(stream);

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if ((h_keys_base.end() == std::find(h_keys_base.begin(),
                                            h_keys_base.end(),
                                            h_keys_test[i])) &&
            !h_accum_or_assigns_test[i])
          expected_size++;
      }
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        bool is_accum = (h_keys_temp[i] == h_keys_test[2]);
        bool is_new_insert =
            (h_keys_test.end() !=
             std::find(h_keys_test.begin(), h_keys_test.end(), h_keys_temp[i]));
        if (is_accum) {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00002));
          }
        } else {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00001));
          }
        }
        if (is_accum || (is_new_insert && (h_keys_temp[i] != h_keys_test[3]))) {
          ASSERT_GE(h_scores_temp[i], start_ts);
          ASSERT_LE(h_scores_temp[i], end_ts);
        } else {
          ASSERT_LE(h_scores_temp[i], start_ts);
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_lfu_basic(size_t max_hbm_for_vectors, int key_start) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 1024;
  constexpr float true_ratio = 0.5;

  TableOptions options;
  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLfu>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  int freq_range = 1000;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, true_ratio);

  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, true_ratio);

  for (int i = 0; i < TEST_TIMES; i++) {
    test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
        h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
        BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF,
        freq_range);

    test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
        h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
        TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
        0xFFFFFFFFFFFFFFFD, freq_range);

    h_accum_or_assigns_base[72] = false;
    h_accum_or_assigns_base[73] = false;

    h_keys_test[2] = h_keys_base[72];
    h_keys_test[3] = h_keys_base[73];

    h_accum_or_assigns_test[2] = true;
    h_accum_or_assigns_test[3] = false;

    h_scores_test[2] = h_keys_base[72] % freq_range;
    h_scores_test[3] = h_keys_base[73] % freq_range;

    for (int i = 0; i < options.dim; i++) {
      h_vectors_test[2 * options.dim + i] =
          h_vectors_base[72 * options.dim + i];
      h_vectors_test[3 * options.dim + i] =
          h_vectors_base[73 * options.dim + i];
    }
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    size_t total_size = 0;
    size_t dump_counter = 0;
    S global_epoch = 1;
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

      table->set_global_epoch(global_epoch);
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        ASSERT_EQ(h_scores_temp[i], h_keys_temp[i] % freq_range);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      global_epoch++;
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

      table->set_global_epoch(global_epoch);
      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if ((h_keys_base.end() == std::find(h_keys_base.begin(),
                                            h_keys_base.end(),
                                            h_keys_test[i])) &&
            !h_accum_or_assigns_test[i])
          expected_size++;
      }
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        bool is_accum = (h_keys_temp[i] == h_keys_test[2]);
        bool is_new_insert =
            (h_keys_test.end() !=
             std::find(h_keys_test.begin(), h_keys_test.end(), h_keys_temp[i]));

        if (is_accum) {
          ASSERT_EQ(h_scores_temp[i], (h_keys_temp[i] % freq_range) * 2);
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00002));
          }
        } else {
          ASSERT_EQ(h_scores_temp[i], (h_keys_temp[i] % freq_range));
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00001));
          }
        }
      }
    }
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_epochlru_basic(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr int RSHIFT_ON_NANO = 20;

  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 128;
  constexpr float true_ratio = 0.5;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kEpochLru>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF);

  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD);

  h_accum_or_assigns_base[72] = false;
  h_accum_or_assigns_base[73] = false;

  h_keys_test[2] = h_keys_base[72];
  h_keys_test[3] = h_keys_base[73];

  h_accum_or_assigns_test[2] = true;
  h_accum_or_assigns_test[3] = false;

  for (int i = 0; i < options.dim; i++) {
    h_vectors_test[2 * options.dim + i] = h_vectors_base[72 * options.dim + i];
    h_vectors_test[3 * options.dim + i] = h_vectors_base[73 * options.dim + i];
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  S global_epoch = 1;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      S start_ts =
          (test_util::host_nano<S>(stream) >> RSHIFT_ON_NANO) & 0xFFFFFFFF;
      table->set_global_epoch(global_epoch);
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      S end_ts =
          (test_util::host_nano<S>(stream) >> RSHIFT_ON_NANO) & 0xFFFFFFFF;

      size_t total_size = table->size(stream);
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        ASSERT_GE(h_scores_temp[i] & 0xFFFFFFFF, start_ts);
        ASSERT_LE(h_scores_temp[i] & 0xFFFFFFFF, end_ts);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      global_epoch++;
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      S start_ts =
          (test_util::host_nano<S>(stream) >> RSHIFT_ON_NANO) & 0xFFFFFFFF;
      table->set_global_epoch(global_epoch);
      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      S end_ts =
          (test_util::host_nano<S>(stream) >> RSHIFT_ON_NANO) & 0xFFFFFFFF;

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if ((h_keys_base.end() == std::find(h_keys_base.begin(),
                                            h_keys_base.end(),
                                            h_keys_test[i])) &&
            !h_accum_or_assigns_test[i])
          expected_size++;
      }
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        bool is_accum = (h_keys_temp[i] == h_keys_test[2]);
        bool is_new_insert =
            (h_keys_test.end() !=
             std::find(h_keys_test.begin(), h_keys_test.end(), h_keys_temp[i]));
        if (is_accum) {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00002));
          }
        } else {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00001));
          }
        }
        if (is_accum || (is_new_insert && (h_keys_temp[i] != h_keys_test[3]))) {
          ASSERT_GE(h_scores_temp[i] & 0xffffffff, start_ts);
          ASSERT_LE(h_scores_temp[i] & 0xffffffff, end_ts);
          ASSERT_EQ(h_scores_temp[i] >> 32 & 0xffffffff, global_epoch);
        } else {
          ASSERT_LE(h_scores_temp[i] & 0xffffffff, start_ts);
          ASSERT_EQ(h_scores_temp[i] >> 32 & 0xffffffff, global_epoch - 1);
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_epochlfu_basic(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 1024;
  constexpr float true_ratio = 0.5;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kEpochLfu>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  int freq_range = 1000;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, true_ratio);

  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, true_ratio);

  test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF,
      freq_range);

  test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD, freq_range);

  h_accum_or_assigns_base[71] = false;
  h_accum_or_assigns_base[72] = false;
  h_accum_or_assigns_base[73] = false;

  // Simulate overflow of low 32bits.
  h_scores_base[71] = static_cast<S>(std::numeric_limits<uint32_t>::max() -
                                     static_cast<uint32_t>(1));

  h_keys_test[1] = h_keys_base[71];
  h_keys_test[2] = h_keys_base[72];
  h_keys_test[3] = h_keys_base[73];

  h_accum_or_assigns_test[1] = true;
  h_accum_or_assigns_test[2] = true;
  h_accum_or_assigns_test[3] = false;

  h_scores_test[1] = h_scores_base[71];
  h_scores_test[2] = h_keys_base[72] % freq_range;
  h_scores_test[3] = h_keys_base[73] % freq_range;

  for (int i = 0; i < options.dim; i++) {
    h_vectors_test[1 * options.dim + i] = h_vectors_base[71 * options.dim + i];
    h_vectors_test[2 * options.dim + i] = h_vectors_base[72 * options.dim + i];
    h_vectors_test[3 * options.dim + i] = h_vectors_base[73 * options.dim + i];
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  S global_epoch = 1;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

      table->set_global_epoch(global_epoch);
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        if (h_keys_temp[i] == h_keys_base[71]) {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, h_scores_base[71]);
          ASSERT_EQ(h_scores_temp[i], expected_score);
        } else {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, (h_keys_temp[i] % freq_range));
          ASSERT_EQ(h_scores_temp[i], expected_score);
        }
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      global_epoch++;
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));

      table->set_global_epoch(global_epoch);
      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if ((h_keys_base.end() == std::find(h_keys_base.begin(),
                                            h_keys_base.end(),
                                            h_keys_test[i])) &&
            !h_accum_or_assigns_test[i])
          expected_size++;
      }
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      ASSERT_TRUE(h_keys_temp.end() != std::find(h_keys_temp.begin(),
                                                 h_keys_temp.end(),
                                                 h_keys_base[71]));

      for (int i = 0; i < dump_counter; i++) {
        bool in_base =
            h_keys_base.end() !=
            std::find(h_keys_base.begin(), h_keys_base.end(), h_keys_temp[i]);

        bool is_accum = (h_keys_temp[i] == h_keys_test[1] ||
                         h_keys_temp[i] == h_keys_test[2]);
        bool is_new_insert =
            (h_keys_test.end() !=
             std::find(h_keys_test.begin(), h_keys_test.end(), h_keys_temp[i]));

        if (is_accum) {
          if (h_keys_temp[i] == h_keys_base[71]) {
            S expected_score = test_util::make_expected_score_for_epochlfu<S>(
                global_epoch, h_scores_base[71] * 2);
            ASSERT_EQ(h_scores_temp[i], expected_score);
          } else {
            S expected_score = test_util::make_expected_score_for_epochlfu<S>(
                global_epoch, (h_keys_temp[i] % freq_range) * 2);
            ASSERT_EQ(h_scores_temp[i], expected_score);
          }
        } else {
          if (h_keys_temp[i] == h_keys_base[71]) {
            S expected_score = test_util::make_expected_score_for_epochlfu<S>(
                global_epoch, h_scores_base[71] * 2);
            ASSERT_EQ(h_scores_temp[i], expected_score);
          } else {
            S expected_score = test_util::make_expected_score_for_epochlfu<S>(
                global_epoch - static_cast<S>(in_base),
                (h_keys_temp[i] % freq_range));

            ASSERT_EQ(h_scores_temp[i], expected_score);
          }
        }
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] *
                                       (is_accum ? 0.00002 : 0.00001)))
              << ",i=" << i << ",is_accum=" << is_accum;
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_customized_basic(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 128;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 128;
  constexpr float true_ratio = 0.3;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;
  std::array<bool, TEMP_KEY_NUM> h_found_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;
  bool* d_found_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_found_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, true_ratio);
  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, true_ratio);

  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF);

  const S base_score_start = 1000;
  for (int i = 0; i < BASE_KEY_NUM; i++) {
    h_scores_base[i] = base_score_start + i;
  }

  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD);
  const S test_score_start = base_score_start + BASE_KEY_NUM;
  for (int i = 0; i < TEST_KEY_NUM; i++) {
    h_scores_test[i] = test_score_start + i;
  }
  for (int i = 64; i < TEST_KEY_NUM; i++) {
    h_keys_test[i] = h_keys_base[i];
    //    h_scores_test[i] = h_scores_base[i];
    for (int j = 0; j < options.dim; j++) {
      h_vectors_test[i * options.dim + j] = h_vectors_base[i * options.dim + j];
    }
  }

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        S expected_score = 0ul;
        bool is_accum = false;
        for (int j = 0; j < BASE_KEY_NUM; j++) {
          if (h_keys_base[j] == h_keys_temp[i]) {
            expected_score = h_scores_base[j];
            is_accum = h_accum_or_assigns_base[j];
          }
        }
        ASSERT_FALSE(is_accum);
        ASSERT_EQ(expected_score, h_scores_temp[i]);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      table->find(TEST_KEY_NUM, d_keys_temp, d_vectors_temp, d_found_temp,
                  nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      CUDA_CHECK(hipMemcpy(h_found_temp.data(), d_found_temp,
                            TEST_KEY_NUM * sizeof(bool),
                            hipMemcpyDeviceToHost));

      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t expected_size = 0;
      for (int i = 0; i < BASE_KEY_NUM; i++) {
        if (!h_accum_or_assigns_base[i]) expected_size++;
      }

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if ((h_keys_base.end() == std::find(h_keys_base.begin(),
                                            h_keys_base.end(),
                                            h_keys_test[i])) &&
            !h_accum_or_assigns_test[i])
          expected_size++;
      }
      expected_size = std::min(expected_size, BUCKET_MAX_SIZE);

      // Some keys in base could be evicted in one operation that allows the
      // same key with `assign` flag in the test can be inserted.
      ASSERT_GE(total_size, expected_size);
      ASSERT_LE(total_size, BUCKET_MAX_SIZE);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, total_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        S expected_score = 888ul;
        S base_score = 888ul;
        S test_score = 888ul;
        bool is_accum_test = false;
        bool is_found_on_base = false;
        for (int j = 0; j < BASE_KEY_NUM; j++) {
          if (h_keys_base[j] == h_keys_temp[i]) {
            is_found_on_base = !h_accum_or_assigns_base[j];
            base_score = h_scores_base[j];
            break;
          }
        }
        for (int j = 0; j < TEST_KEY_NUM; j++) {
          if (h_keys_test[j] == h_keys_temp[i]) {
            is_accum_test = h_accum_or_assigns_test[j];
            test_score = h_scores_test[j];
            break;
          }
        }
        if (is_found_on_base && is_accum_test) expected_score = test_score;
        if (is_found_on_base && !is_accum_test) expected_score = base_score;
        if (!is_found_on_base && is_accum_test) assert(false);
        if (!is_found_on_base && !is_accum_test) expected_score = test_score;

        // Some keys in base could be evicted in one operation that allows the
        // same key with `assign` flag in the test can be inserted.
        ASSERT_EQ(expected_score, h_scores_temp[i])
            << " " << is_found_on_base << " " << is_accum_test << " "
            << base_score << " " << test_score;
        if (is_found_on_base && is_accum_test) {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00002));
          }
        } else {
          for (int j = 0; j < options.dim; j++) {
            ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                      static_cast<float>(h_keys_temp[i] * 0.00001));
          }
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));
  CUDA_CHECK(hipFree(d_found_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_customized_advanced(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 8;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 256;
  constexpr float base_true_ratio = 0.0f;
  constexpr float test_true_ratio = 0.5f;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;
  std::array<bool, BASE_KEY_NUM> h_accum_or_assigns_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;
  std::array<bool, TEST_KEY_NUM> h_accum_or_assigns_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, TEMP_KEY_NUM * sizeof(bool)));

  test_util::create_random_bools<K>(h_accum_or_assigns_base.data(),
                                    BASE_KEY_NUM, base_true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF);

  const S base_score_start = 1000;
  for (int i = 0; i < BASE_KEY_NUM; i++) {
    h_scores_base[i] = base_score_start + i;
  }

  test_util::create_random_bools<K>(h_accum_or_assigns_test.data(),
                                    TEST_KEY_NUM, test_true_ratio);
  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD);

  h_keys_test[4] = h_keys_base[72];
  h_keys_test[5] = h_keys_base[73];
  h_keys_test[6] = h_keys_base[74];
  h_keys_test[7] = h_keys_base[75];

  h_accum_or_assigns_base[72] = false;
  h_accum_or_assigns_base[73] = false;
  h_accum_or_assigns_base[74] = false;
  h_accum_or_assigns_base[75] = false;
  // replace four new keys to lower scores, would not be inserted.
  h_scores_test[0] = 20;
  h_scores_test[1] = 78;
  h_scores_test[2] = 97;
  h_scores_test[3] = 98;

  // replace three exist keys to new scores, just refresh the score for them.
  h_scores_test[4] = 99;
  h_scores_test[5] = 1010;
  h_scores_test[6] = 1020;
  h_scores_test[7] = 1035;

  h_accum_or_assigns_test[0] = false;
  h_accum_or_assigns_test[1] = false;
  h_accum_or_assigns_test[2] = false;
  h_accum_or_assigns_test[3] = false;

  h_accum_or_assigns_test[4] = true;
  h_accum_or_assigns_test[5] = true;
  h_accum_or_assigns_test[6] = true;
  h_accum_or_assigns_test[7] = false;

  for (int i = 4; i < TEST_KEY_NUM; i++) {
    for (int j = 0; j < options.dim; j++) {
      h_vectors_test[i * options.dim + j] =
          static_cast<V>(h_keys_test[i] * 0.00001);
    }
  }

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base.data(),
                     BASE_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      table->accum_or_assign(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t expected_size = 0;
      for (const auto accum : h_accum_or_assigns_base) {
        if (!accum) expected_size++;
      }

      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      std::array<S, BASE_KEY_NUM> h_scores_temp_sorted(h_scores_temp);
      std::sort(h_scores_temp_sorted.begin(), h_scores_temp_sorted.end());

      for (int i = 0; i < dump_counter; i++) {
        S expected_score = 0ul;
        for (int j = 0; j < BASE_KEY_NUM; j++) {
          if (h_keys_temp[i] == h_keys_base[j]) {
            expected_score = h_scores_base[j];
            break;
          }
        }
        ASSERT_EQ(h_scores_temp[i], expected_score);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      std::unordered_map<K, bool> base_found_map;
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(
          hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_test.data(),
                     TEST_KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));

      table->accum_or_assign(TEST_KEY_NUM, d_keys_temp, d_vectors_temp,
                             d_accum_or_assigns_temp, d_scores_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t expected_size = 0;
      for (const auto accum : h_accum_or_assigns_base) {
        if (!accum) expected_size++;
      }
      expected_size = std::max(expected_size, BUCKET_MAX_SIZE);
      ASSERT_EQ(total_size, expected_size);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, expected_size);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < TEST_KEY_NUM; i++) {
        if (i < 4) {
          ASSERT_EQ(h_keys_temp.end(),
                    std::find(h_keys_temp.begin(), h_keys_temp.end(),
                              h_keys_test[i]));
        } else {
          ASSERT_NE(h_keys_temp.end(),
                    std::find(h_keys_temp.begin(), h_keys_temp.end(),
                              h_keys_test[i]));
        }
      }
      for (int i = 0; i < TEMP_KEY_NUM; i++) {
        if (h_keys_temp[i] == h_keys_test[4])
          ASSERT_EQ(h_scores_temp[i], h_scores_test[4]);
        if (h_keys_temp[i] == h_keys_test[5])
          ASSERT_EQ(h_scores_temp[i], h_scores_test[5]);
        if (h_keys_temp[i] == h_keys_test[6])
          ASSERT_EQ(h_scores_temp[i], h_scores_test[6]);
        if (h_keys_temp[i] == h_keys_test[7])
          ASSERT_NE(h_scores_temp[i], h_scores_test[7]);
        bool is_accum =
            (h_keys_temp[i] != h_keys_test[7]) &&
            (h_keys_test.end() != std::find(h_keys_test.begin() + 4,
                                            h_keys_test.end(), h_keys_temp[i]));
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] *
                                       (is_accum ? 0.00002 : 0.00001)));
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_customized_correct_rate(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr uint64_t BATCH_SIZE = 1024 * 1024ul;
  constexpr uint64_t STEPS = 128;
  constexpr uint64_t MAX_BUCKET_SIZE = 128;
  constexpr uint64_t INIT_CAPACITY = BATCH_SIZE * STEPS;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t TEST_TIMES = 1;
  float expected_correct_rate = 0.964;
  const int rounds = 3;
  constexpr float true_ratio = 0.0;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_bucket_size = MAX_BUCKET_SIZE;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kCustomized>;

  K* h_keys_base = test_util::HostBuffer<K>(BATCH_SIZE).ptr();
  S* h_scores_base = test_util::HostBuffer<S>(BATCH_SIZE).ptr();
  V* h_vectors_base = test_util::HostBuffer<V>(BATCH_SIZE * options.dim).ptr();
  bool* h_accum_or_assigns_base = test_util::HostBuffer<bool>(BATCH_SIZE).ptr();

  K* h_keys_temp = test_util::HostBuffer<K>(MAX_CAPACITY).ptr();
  S* h_scores_temp = test_util::HostBuffer<S>(MAX_CAPACITY).ptr();
  V* h_vectors_temp =
      test_util::HostBuffer<V>(MAX_CAPACITY * options.dim).ptr();

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;
  bool* d_accum_or_assigns_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, MAX_CAPACITY * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, MAX_CAPACITY * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, MAX_CAPACITY * sizeof(V) * options.dim));
  CUDA_CHECK(hipMalloc(&d_accum_or_assigns_temp, MAX_CAPACITY * sizeof(bool)));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t global_start_key = 100000;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);
    size_t start_key = global_start_key;

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    for (int r = 0; r < rounds; r++) {
      size_t expected_min_key = global_start_key + INIT_CAPACITY * r;
      size_t expected_max_key = global_start_key + INIT_CAPACITY * (r + 1) - 1;
      size_t expected_table_size =
          (r == 0) ? size_t(expected_correct_rate * INIT_CAPACITY)
                   : INIT_CAPACITY;

      for (int s = 0; s < STEPS; s++) {
        test_util::create_random_bools<K>(h_accum_or_assigns_base, BATCH_SIZE,
                                          true_ratio);
        test_util::create_continuous_keys<K, S, V, DIM>(
            h_keys_base, h_scores_base, h_vectors_base, BATCH_SIZE, start_key);
        start_key += BATCH_SIZE;

        CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base, BATCH_SIZE * sizeof(K),
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base,
                              BATCH_SIZE * sizeof(S), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base,
                              BATCH_SIZE * sizeof(V) * options.dim,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_accum_or_assigns_temp, h_accum_or_assigns_base,
                              BATCH_SIZE * sizeof(bool),
                              hipMemcpyHostToDevice));
        table->accum_or_assign(BATCH_SIZE, d_keys_temp, d_vectors_temp,
                               d_accum_or_assigns_temp, d_scores_temp, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));
      }

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_GE(total_size, expected_table_size);
      ASSERT_EQ(MAX_CAPACITY, table->capacity());

      size_t dump_counter = table->export_batch(
          MAX_CAPACITY, 0, d_keys_temp, d_vectors_temp, d_scores_temp, stream);

      CUDA_CHECK(hipMemcpy(h_keys_temp, d_keys_temp, MAX_CAPACITY * sizeof(K),
                            hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(h_scores_temp, d_scores_temp,
                            MAX_CAPACITY * sizeof(S), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(h_vectors_temp, d_vectors_temp,
                            MAX_CAPACITY * sizeof(V) * options.dim,
                            hipMemcpyDeviceToHost));

      size_t bigger_score_counter = 0;
      K max_key = 0;
      size_t values_error_counter = 0;
      for (int i = 0; i < dump_counter; i++) {
        ASSERT_EQ(h_keys_temp[i], h_scores_temp[i]);
        max_key = std::max(max_key, h_keys_temp[i]);
        if (h_scores_temp[i] >= expected_min_key) bigger_score_counter++;
        for (int j = 0; j < options.dim; j++) {
          if (h_vectors_temp[i * options.dim + j] !=
              static_cast<float>(h_keys_temp[i] * 0.00001)) {
            values_error_counter++;
          }
        }
      }

      ASSERT_EQ(values_error_counter, 0);
      float correct_rate = (bigger_score_counter * 1.0) / MAX_CAPACITY;
      std::cout << std::setprecision(3) << "[Round " << r << "]"
                << "correct_rate=" << correct_rate << std::endl;
      ASSERT_GE(max_key, expected_max_key);
      ASSERT_GE(correct_rate, expected_correct_rate);
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));
  CUDA_CHECK(hipFree(d_accum_or_assigns_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_dynamic_rehash_on_multi_threads(size_t max_hbm_for_vectors, int key_start = 0) {
  constexpr uint64_t BUCKET_MAX_SIZE = 128ul;
  constexpr uint64_t INIT_CAPACITY = 4 * 1024 - BUCKET_MAX_SIZE - 1;
  constexpr uint64_t MAX_CAPACITY = 16 * 1024 * INIT_CAPACITY;
  constexpr uint64_t KEY_NUM = 256;
  constexpr uint64_t THREAD_N = 8;

  std::vector<std::thread> threads;

  TableOptions options;

  options.reserved_key_start_bit = key_start;
  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_load_factor = 0.50f;
  options.max_bucket_size = BUCKET_MAX_SIZE;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;

  std::shared_ptr<Table> table = std::make_shared<Table>();
  table->init(options);
  ASSERT_EQ(table->bucket_count(), 32);

  auto worker_function = [&table, KEY_NUM, options](int task_n) {
    constexpr float true_ratio = 0.5;
    K* h_keys;
    V* h_vectors;
    bool* h_found;
    bool* h_accum_or_assigns;
    K* h_keys_temp;
    V* h_vectors_temp;
    bool* h_found_temp;

    size_t current_capacity = table->capacity();

    CUDA_CHECK(hipHostMalloc(&h_keys, KEY_NUM * sizeof(K)));
    CUDA_CHECK(hipHostMalloc(&h_vectors, KEY_NUM * sizeof(V) * options.dim));
    CUDA_CHECK(hipHostMalloc(&h_keys_temp, KEY_NUM * sizeof(K)));
    CUDA_CHECK(
        hipHostMalloc(&h_vectors_temp, KEY_NUM * sizeof(V) * options.dim));
    CUDA_CHECK(hipHostMalloc(&h_found_temp, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipHostMalloc(&h_found, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipHostMalloc(&h_accum_or_assigns, KEY_NUM * sizeof(bool)));

    K* d_keys;
    V* d_vectors;
    bool* d_found;
    bool* d_accum_or_assigns;

    CUDA_CHECK(hipMalloc(&d_keys, KEY_NUM * sizeof(K)));
    CUDA_CHECK(hipMalloc(&d_vectors, KEY_NUM * sizeof(V) * options.dim));
    CUDA_CHECK(hipMalloc(&d_found, KEY_NUM * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_accum_or_assigns, KEY_NUM * sizeof(bool)));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    while (table->capacity() * 2 < MAX_CAPACITY) {
      test_util::create_random_bools<K>(h_accum_or_assigns, KEY_NUM,
                                        true_ratio);

      test_util::create_random_keys<K, S, V, DIM>(h_keys, nullptr, h_vectors,
                                                  KEY_NUM);
      CUDA_CHECK(hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K),
                            hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors, h_vectors,
                            KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_accum_or_assigns, h_accum_or_assigns,
                            KEY_NUM * sizeof(bool), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemset(d_found, 0, KEY_NUM * sizeof(bool)));

      table->find(KEY_NUM, d_keys, d_vectors, d_found, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      CUDA_CHECK(hipMemcpy(h_keys_temp, d_keys, KEY_NUM * sizeof(K),
                            hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(h_vectors_temp, d_vectors,
                            KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(h_found_temp, d_found, KEY_NUM * sizeof(bool),
                            hipMemcpyDeviceToHost));

      table->accum_or_assign(KEY_NUM, d_keys, d_vectors, d_accum_or_assigns,
                             nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      CUDA_CHECK(hipMemset(d_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
      table->find(KEY_NUM, d_keys, d_vectors, d_found, nullptr, stream);

      CUDA_CHECK(hipStreamSynchronize(stream));
      int found_num = 0;
      size_t expected_size = 0;
      for (int i = 0; i < KEY_NUM; i++) {
        if (h_found_temp[i] || !h_accum_or_assigns[i]) expected_size++;
      }

      CUDA_CHECK(hipMemset(h_found, 0, KEY_NUM * sizeof(bool)));
      CUDA_CHECK(hipMemset(h_vectors, 0, KEY_NUM * sizeof(V) * options.dim));
      CUDA_CHECK(hipMemcpy(h_keys, d_keys, KEY_NUM * sizeof(K),
                            hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
                            hipMemcpyDeviceToHost));

      CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                            KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDeviceToHost));
      for (int i = 0; i < KEY_NUM; i++) {
        if (h_found[i]) {
          found_num++;
          for (int j = 0; j < options.dim; j++) {
            if (h_found_temp[i] && h_accum_or_assigns[i]) {
              ASSERT_EQ(h_vectors[i * options.dim + j],
                        h_vectors_temp[i * options.dim + j] +
                            static_cast<float>(h_keys[i] * 0.00001));
            } else {
              ASSERT_EQ(h_vectors[i * options.dim + j],
                        static_cast<float>(h_keys[i] * 0.00001));
            }
          }
        }
      }
      ASSERT_EQ(found_num, expected_size);
      if (task_n == 0 && current_capacity != table->capacity()) {
        std::cout << "[test_dynamic_rehash_on_multi_threads] The capacity "
                     "changed from "
                  << current_capacity << " to " << table->capacity()
                  << std::endl;
        current_capacity = table->capacity();
      }
      CUDA_CHECK(hipStreamSynchronize(stream));
    }
    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipMemcpy(h_vectors, d_vectors,
                          KEY_NUM * sizeof(V) * options.dim,
                          hipMemcpyDeviceToHost));

    CUDA_CHECK(hipHostFree(h_keys));
    CUDA_CHECK(hipHostFree(h_found));
    CUDA_CHECK(hipHostFree(h_vectors));
    CUDA_CHECK(hipHostFree(h_keys_temp));
    CUDA_CHECK(hipHostFree(h_found_temp));
    CUDA_CHECK(hipHostFree(h_vectors_temp));
    CUDA_CHECK(hipHostFree(h_accum_or_assigns));

    CUDA_CHECK(hipFree(d_keys));
    CUDA_CHECK(hipFree(d_vectors));
    CUDA_CHECK(hipFree(d_found));
    CUDA_CHECK(hipFree(d_accum_or_assigns));
    CUDA_CHECK(hipDeviceSynchronize());

    CudaCheckError();
  };

  for (int i = 0; i < THREAD_N; ++i)
    threads.emplace_back(std::thread(worker_function, i));

  for (auto& th : threads) {
    th.join();
  }
  ASSERT_GE(table->capacity() * 2, MAX_CAPACITY);
}

template <typename K, typename V, typename S, typename Table, size_t dim = 64>
void CheckAccumOrAssignValues(Table* table,
                              test_util::KVMSBuffer<K, V, S>& data_buffer,
                              size_t len, hipStream_t stream) {
  std::map<K, test_util::ValueArray<V, dim>> map_before_insert;
  std::map<K, test_util::ValueArray<V, dim>> map_after_insert;
  std::unordered_set<K> map_current_batch;

  K* keys = data_buffer.keys_ptr();
  V* values = data_buffer.values_ptr();
  S* scores = data_buffer.scores_ptr();

  for (int i = 0; i < len; i++) {
    map_current_batch.insert(data_buffer.keys_ptr(false)[i]);
  }

  K* h_tmp_keys = nullptr;
  V* h_tmp_values = nullptr;
  S* h_tmp_scores = nullptr;
  bool* h_tmp_accum_or_assigns = nullptr;

  K* d_tmp_keys = nullptr;
  V* d_tmp_values = nullptr;
  S* d_tmp_scores = nullptr;
  bool* d_tmp_accum_or_assigns = nullptr;

  size_t table_size_before = table->size(stream);
  size_t cap = table_size_before + len;

  h_tmp_keys = (K*)malloc(cap * sizeof(K));
  h_tmp_values = (V*)malloc(cap * dim * sizeof(V));
  h_tmp_scores = (S*)malloc(cap * sizeof(S));
  h_tmp_accum_or_assigns = (bool*)malloc(cap * sizeof(bool));

  CUDA_CHECK(hipMallocAsync(&d_tmp_keys, cap * sizeof(K), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_keys, 0, cap * sizeof(K), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_values, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_values, 0, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_scores, cap * sizeof(S), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_scores, 0, cap * sizeof(S), stream));
  CUDA_CHECK(
      hipMallocAsync(&d_tmp_accum_or_assigns, cap * sizeof(bool), stream));
  CUDA_CHECK(
      hipMemsetAsync(d_tmp_accum_or_assigns, 0, cap * sizeof(bool), stream));

  table->find(len, keys, d_tmp_values, d_tmp_accum_or_assigns, nullptr, stream);
  CUDA_CHECK(hipMemcpyAsync(h_tmp_accum_or_assigns, d_tmp_accum_or_assigns,
                             len * sizeof(bool), hipMemcpyDeviceToHost,
                             stream));

  CUDA_CHECK(hipMemsetAsync(d_tmp_keys, 0, cap * sizeof(K), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_values, 0, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_scores, 0, cap * sizeof(S), stream));

  size_t table_size_verify0 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_scores, stream);
  ASSERT_EQ(table_size_before, table_size_verify0);

  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_before * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_before * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores, d_tmp_scores,
                             table_size_before * sizeof(S),
                             hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  for (size_t i = 0; i < table_size_verify0; i++) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    map_before_insert[h_tmp_keys[i]] = *vec;
  }

  auto start = std::chrono::steady_clock::now();
  table->accum_or_assign(len, keys, values, d_tmp_accum_or_assigns, nullptr,
                         stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  auto end = std::chrono::steady_clock::now();
  auto diff = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

  float dur = diff.count();

  size_t table_size_after = table->size(stream);
  size_t table_size_verify1 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_scores, stream);

  ASSERT_EQ(table_size_verify1, table_size_after);

  size_t new_cap = table_size_after;
  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_after * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_after * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores, d_tmp_scores,
                             table_size_after * sizeof(S),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_accum_or_assigns, d_tmp_accum_or_assigns,
                             table_size_after * sizeof(bool),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));
  int64_t new_cap_K = (int64_t)new_cap;
  for (int64_t i = new_cap_K - 1; i >= 0; i--) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    map_after_insert[h_tmp_keys[i]] = *vec;
  }

  size_t value_diff_cnt = 0;
  for (auto& it : map_after_insert) {
    test_util::ValueArray<V, dim>& vec = map_after_insert.at(it.first);
    bool existed =
        (map_before_insert.end() != map_before_insert.find(it.first));
    bool required =
        (map_current_batch.end() != map_current_batch.find(it.first));
    V expected_value = 0;

    if (existed) {
      if (required) {
        expected_value = (map_before_insert.at(it.first)[0] +
                          static_cast<V>(it.first * 0.00001));
      } else {
        expected_value = map_before_insert.at(it.first)[0];
      }
    } else {
      if (required) {
        expected_value = static_cast<V>(it.first * 0.00001);
      }
    }
    for (size_t j = 0; j < dim; j++) {
      if (vec[j] != expected_value) {
        ++value_diff_cnt;
        break;
      }
    }
  }
  ASSERT_EQ(value_diff_cnt, 0);
  std::cout << "Check accum_or_assign behavior got "
            << "value_diff_cnt: " << value_diff_cnt
            << ", while table_size_before: " << table_size_before
            << ", while table_size_after: " << table_size_after
            << ", while len: " << len << std::endl;

  CUDA_CHECK(hipFreeAsync(d_tmp_keys, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_values, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_scores, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_accum_or_assigns, stream));
  free(h_tmp_keys);
  free(h_tmp_values);
  free(h_tmp_scores);
  free(h_tmp_accum_or_assigns);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

void test_accum_or_assign_values_check(size_t max_hbm_for_vectors) {
  const size_t U = 524288;
  const size_t init_capacity = 1024;
  const size_t B = 524288 + 13;
  constexpr size_t dim = 64;

  TableOptions opt;

  opt.max_capacity = U;
  opt.init_capacity = init_capacity;
  opt.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;
  opt.dim = dim;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(opt);

  test_util::KVMSBuffer<K, V, S> data_buffer;
  data_buffer.Reserve(B, dim, stream);

  size_t offset = 0;
  S score = 0;
  for (int i = 0; i < 20; i++) {
    test_util::create_random_keys<K, S, V, dim>(
        data_buffer.keys_ptr(false), data_buffer.scores_ptr(false),
        data_buffer.values_ptr(false), (int)B, B * 16);
    data_buffer.SyncData(true, stream);

    CheckAccumOrAssignValues<K, V, S, Table, dim>(table.get(), data_buffer, B,
                                                  stream);

    offset += B;
    score += 1;
  }
}

TEST(AccumOrAssignTest, test_export_batch_if) {
  test_export_batch_if(16, 22);
  test_export_batch_if(0, 0);
}
TEST(AccumOrAssignTest, test_basic_when_full) {
  test_basic_when_full(16, 2);
  test_basic_when_full(0, 0);
}
TEST(AccumOrAssignTest, test_erase_if_pred) {
  test_erase_if_pred(16, 0);
  test_erase_if_pred(0, 5);
}
TEST(AccumOrAssignTest, test_rehash) {
  test_rehash(16, 7);
  test_rehash(0, 0);
}
TEST(AccumOrAssignTest, test_rehash_on_big_batch) {
  test_rehash_on_big_batch(16, 9);
  test_rehash_on_big_batch(0, 0);
}
TEST(AccumOrAssignTest, test_dynamic_rehash_on_multi_threads) {
  test_dynamic_rehash_on_multi_threads(16, 56);
  test_dynamic_rehash_on_multi_threads(0);
}
TEST(AccumOrAssignTest, test_evict_strategy_lru_basic) {
  test_evict_strategy_lru_basic(16);
  test_evict_strategy_lru_basic(0);
}
TEST(AccumOrAssignTest, test_evict_strategy_lfu_basic) {
  test_evict_strategy_lfu_basic(16, 3);
  // TODO(rhdong): Add back when diff error issue fixed in hybrid mode.
  // test_evict_strategy_lfu_basic(0);
}

TEST(AccumOrAssignTest, test_evict_strategy_epochlru_basic) {
  test_evict_strategy_epochlru_basic(16, 33);
  test_evict_strategy_epochlru_basic(0);
}

TEST(AccumOrAssignTest, test_evict_strategy_epochlfu_basic) {
  test_evict_strategy_epochlfu_basic(16);
  test_evict_strategy_epochlfu_basic(0, 44);
}
TEST(AccumOrAssignTest, test_evict_strategy_customized_basic) {
  test_evict_strategy_customized_basic(16);
  test_evict_strategy_customized_basic(0, 23);
}
TEST(AccumOrAssignTest, test_evict_strategy_customized_advanced) {
  test_evict_strategy_customized_advanced(16, 16);
  test_evict_strategy_customized_advanced(0);
}
TEST(AccumOrAssignTest, test_evict_strategy_customized_correct_rate) {
  // TODO(rhdong): after blossom CI issue is resolved, the skip logic.
  const bool skip_hmem_check = (nullptr != std::getenv("IS_BLOSSOM_CI"));
  test_evict_strategy_customized_correct_rate(16, 61);
  if (!skip_hmem_check) {
    test_evict_strategy_customized_correct_rate(0);
  } else {
    std::cout << "The HMEM check is skipped in blossom CI!" << std::endl;
  }
}

TEST(AccumOrAssignTest, test_accum_or_assign_values_check) {
  test_accum_or_assign_values_check(16);
  // TODO(rhdong): Add back when diff error issue fixed in hybrid mode.
  // test_insert_or_assign_values_check(0);
}