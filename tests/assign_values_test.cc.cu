#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * test API: assign_values
 */

#include <gtest/gtest.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <array>
#include <iostream>
#include <thread>
#include <vector>
#include "merlin_hashtable.cuh"
#include "test_util.cuh"

constexpr size_t DIM = 16;
using K = uint64_t;
using V = float;
using S = uint64_t;
using EvictStrategy = nv::merlin::EvictStrategy;
using TableOptions = nv::merlin::HashTableOptions;

void test_evict_strategy_lru_basic(size_t max_hbm_for_vectors) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 128;

  TableOptions options;

  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF);

  test_util::create_keys_in_one_buckets<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD);

  h_keys_test[2] = h_keys_base[72];
  h_keys_test[3] = h_keys_base[73];

  for (int i = 0; i < options.dim; i++) {
    h_vectors_test[2 * options.dim + i] =
        static_cast<float>(h_keys_base[72] * 0.00002);
    h_vectors_test[3 * options.dim + i] =
        static_cast<float>(h_keys_base[73] * 0.00002);
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      S start_ts = test_util::host_nano<S>(stream);
      table->find_or_insert(BASE_KEY_NUM, d_keys_temp, d_vectors_temp, nullptr,
                            stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      S end_ts = test_util::host_nano<S>(stream);

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, BUCKET_MAX_SIZE);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, BUCKET_MAX_SIZE);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      std::array<S, BASE_KEY_NUM> h_scores_temp_sorted(h_scores_temp);
      std::sort(h_scores_temp_sorted.begin(), h_scores_temp_sorted.end());

      ASSERT_GE(h_scores_temp_sorted[0], start_ts);
      ASSERT_LE(h_scores_temp_sorted[TEST_KEY_NUM - 1], end_ts);
      for (int i = 0; i < dump_counter; i++) {
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));

      S start_ts = test_util::host_nano<S>(stream);
      table->assign_values(TEST_KEY_NUM, d_keys_temp, d_vectors_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, BUCKET_MAX_SIZE);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, BUCKET_MAX_SIZE);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < TEMP_KEY_NUM; i++) {
        V expected_v = (h_keys_temp[i] == h_keys_test[2] ||
                        h_keys_temp[i] == h_keys_test[3])
                           ? static_cast<V>(h_keys_temp[i] * 0.00002)
                           : static_cast<V>(h_keys_temp[i] * 0.00001);
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j], expected_v);
        }
        ASSERT_LE(h_scores_temp[i], start_ts);
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

void test_evict_strategy_epochlfu_basic(size_t max_hbm_for_vectors) {
  constexpr uint64_t BUCKET_NUM = 8UL;
  constexpr uint64_t BUCKET_MAX_SIZE = 128UL;
  constexpr uint64_t INIT_CAPACITY = BUCKET_NUM * BUCKET_MAX_SIZE;  // 1024UL;
  constexpr uint64_t MAX_CAPACITY = INIT_CAPACITY;
  constexpr uint64_t BASE_KEY_NUM = BUCKET_MAX_SIZE;
  constexpr uint64_t TEST_KEY_NUM = 4;
  constexpr uint64_t TEMP_KEY_NUM = std::max(BASE_KEY_NUM, TEST_KEY_NUM);
  constexpr uint64_t TEST_TIMES = 128;

  TableOptions options;

  options.init_capacity = INIT_CAPACITY;
  options.max_capacity = MAX_CAPACITY;
  options.dim = DIM;
  options.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kEpochLfu>;

  std::array<K, BASE_KEY_NUM> h_keys_base;
  std::array<S, BASE_KEY_NUM> h_scores_base;
  std::array<V, BASE_KEY_NUM * DIM> h_vectors_base;

  std::array<K, TEST_KEY_NUM> h_keys_test;
  std::array<S, TEST_KEY_NUM> h_scores_test;
  std::array<V, TEST_KEY_NUM * DIM> h_vectors_test;

  std::array<K, TEMP_KEY_NUM> h_keys_temp;
  std::array<S, TEMP_KEY_NUM> h_scores_temp;
  std::array<V, TEMP_KEY_NUM * DIM> h_vectors_temp;

  K* d_keys_temp;
  S* d_scores_temp = nullptr;
  V* d_vectors_temp;

  int freq_range = 1000;

  CUDA_CHECK(hipMalloc(&d_keys_temp, TEMP_KEY_NUM * sizeof(K)));
  CUDA_CHECK(hipMalloc(&d_scores_temp, TEMP_KEY_NUM * sizeof(S)));
  CUDA_CHECK(
      hipMalloc(&d_vectors_temp, TEMP_KEY_NUM * sizeof(V) * options.dim));

  test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
      h_keys_base.data(), h_scores_base.data(), h_vectors_base.data(),
      BASE_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0, 0x3FFFFFFFFFFFFFFF,
      freq_range);

  test_util::create_keys_in_one_buckets_lfu<K, S, V, DIM>(
      h_keys_test.data(), h_scores_test.data(), h_vectors_test.data(),
      TEST_KEY_NUM, INIT_CAPACITY, BUCKET_MAX_SIZE, 1, 0x3FFFFFFFFFFFFFFF,
      0xFFFFFFFFFFFFFFFD, freq_range);

  // Simulate overflow of low 32bits.
  h_scores_base[71] = static_cast<S>(std::numeric_limits<uint32_t>::max() -
                                     static_cast<uint32_t>(1));

  h_keys_test[1] = h_keys_base[71];
  h_keys_test[2] = h_keys_base[72];
  h_keys_test[3] = h_keys_base[73];

  h_scores_test[1] = h_scores_base[71];
  h_scores_test[2] = h_keys_base[72] % freq_range;
  h_scores_test[3] = h_keys_base[73] % freq_range;

  for (int i = 0; i < options.dim; i++) {
    h_vectors_test[1 * options.dim + i] =
        static_cast<float>(h_keys_base[71] * 0.00002);
    h_vectors_test[2 * options.dim + i] =
        static_cast<float>(h_keys_base[72] * 0.00002);
    h_vectors_test[3 * options.dim + i] =
        static_cast<float>(h_keys_base[73] * 0.00002);
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  size_t total_size = 0;
  size_t dump_counter = 0;
  S global_epoch = 1;
  for (int i = 0; i < TEST_TIMES; i++) {
    std::unique_ptr<Table> table = std::make_unique<Table>();
    table->init(options);

    total_size = table->size(stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(total_size, 0);

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_base.data(),
                            BASE_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_base.data(),
                            BASE_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_base.data(),
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      EvictStrategy::set_global_epoch(global_epoch);
      table->find_or_insert(BASE_KEY_NUM, d_keys_temp, d_vectors_temp,
                            d_scores_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, BUCKET_MAX_SIZE);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, BUCKET_MAX_SIZE);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            BASE_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            BASE_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            BASE_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      for (int i = 0; i < dump_counter; i++) {
        if (h_keys_temp[i] == h_keys_base[71]) {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, h_scores_base[71]);
          ASSERT_EQ(h_scores_temp[i], expected_score);
        } else {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, (h_keys_temp[i] % freq_range));
          ASSERT_EQ(h_scores_temp[i], expected_score);
        }
        for (int j = 0; j < options.dim; j++) {
          ASSERT_EQ(h_vectors_temp[i * options.dim + j],
                    static_cast<float>(h_keys_temp[i] * 0.00001));
        }
      }
    }

    {
      CUDA_CHECK(hipMemcpy(d_keys_temp, h_keys_test.data(),
                            TEST_KEY_NUM * sizeof(K), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_scores_temp, h_scores_test.data(),
                            TEST_KEY_NUM * sizeof(S), hipMemcpyHostToDevice));
      CUDA_CHECK(hipMemcpy(d_vectors_temp, h_vectors_test.data(),
                            TEST_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyHostToDevice));
      table->assign_values(TEST_KEY_NUM, d_keys_temp, d_vectors_temp, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));

      size_t total_size = table->size(stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      ASSERT_EQ(total_size, BUCKET_MAX_SIZE);

      dump_counter = table->export_batch(table->capacity(), 0, d_keys_temp,
                                         d_vectors_temp, d_scores_temp, stream);
      ASSERT_EQ(dump_counter, BUCKET_MAX_SIZE);

      CUDA_CHECK(hipMemcpy(h_keys_temp.data(), d_keys_temp,
                            TEMP_KEY_NUM * sizeof(K), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_scores_temp.data(), d_scores_temp,
                            TEMP_KEY_NUM * sizeof(S), hipMemcpyDefault));
      CUDA_CHECK(hipMemcpy(h_vectors_temp.data(), d_vectors_temp,
                            TEMP_KEY_NUM * sizeof(V) * options.dim,
                            hipMemcpyDefault));

      ASSERT_TRUE(h_keys_temp.end() != std::find(h_keys_temp.begin(),
                                                 h_keys_temp.end(),
                                                 h_keys_base[71]));

      for (int i = 0; i < dump_counter; i++) {
        if (h_keys_temp[i] == h_keys_base[71]) {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, h_scores_base[71]);
          ASSERT_EQ(h_scores_temp[i], expected_score);
        } else {
          S expected_score = test_util::make_expected_score_for_epochlfu<S>(
              global_epoch, (h_keys_temp[i] % freq_range));
          ASSERT_EQ(h_scores_temp[i], expected_score);
        }
        for (int j = 0; j < options.dim; j++) {
          V expected_v = (h_keys_temp[i] == h_keys_test[1] ||
                          h_keys_temp[i] == h_keys_test[2] ||
                          h_keys_temp[i] == h_keys_test[3])
                             ? static_cast<V>(h_keys_temp[i] * 0.00002)
                             : static_cast<V>(h_keys_temp[i] * 0.00001);
          ASSERT_EQ(h_vectors_temp[i * options.dim + j], expected_v);
        }
      }
    }
  }
  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipFree(d_keys_temp));
  CUDA_CHECK(hipFree(d_scores_temp));
  CUDA_CHECK(hipFree(d_vectors_temp));

  CUDA_CHECK(hipDeviceSynchronize());

  CudaCheckError();
}

template <typename K, typename V, typename S, typename Table, size_t dim = 64>
void CheckAssignOnEpochLfu(Table* table,
                           test_util::KVMSBuffer<K, V, S>* data_buffer,
                           test_util::KVMSBuffer<K, V, S>* evict_buffer,
                           test_util::KVMSBuffer<K, V, S>* pre_data_buffer,
                           size_t len, hipStream_t stream, TableOptions& opt,
                           unsigned int global_epoch) {
  std::map<K, test_util::ValueArray<V, dim>> values_map_before_insert;
  std::map<K, test_util::ValueArray<V, dim>> values_map_after_insert;

  std::unordered_map<K, S> scores_map_before_insert;
  std::map<K, S> scores_map_after_insert;

  std::map<K, S> scores_map_current_batch;
  std::map<K, S> scores_map_current_evict;

  K* keys = data_buffer->keys_ptr();
  V* values = data_buffer->values_ptr();
  S* scores = data_buffer->scores_ptr();

  K* evicted_keys = evict_buffer->keys_ptr();
  V* evicted_values = evict_buffer->values_ptr();
  S* evicted_scores = evict_buffer->scores_ptr();

  for (size_t i = 0; i < len; i++) {
    scores_map_current_batch[data_buffer->keys_ptr(false)[i]] =
        data_buffer->scores_ptr(false)[i];
  }

  K* h_tmp_keys = nullptr;
  V* h_tmp_values = nullptr;
  S* h_tmp_scores = nullptr;
  bool* h_tmp_founds = nullptr;

  K* d_tmp_keys = nullptr;
  V* d_tmp_values = nullptr;
  S* d_tmp_scores = nullptr;
  bool* d_tmp_founds = nullptr;

  size_t table_size_before = table->size(stream);
  size_t cap = table_size_before + len;

  CUDA_CHECK(hipMallocAsync(&d_tmp_keys, cap * sizeof(K), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_keys, 0, cap * sizeof(K), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_values, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_values, 0, cap * dim * sizeof(V), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_scores, cap * sizeof(S), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_scores, 0, cap * sizeof(S), stream));
  CUDA_CHECK(hipMallocAsync(&d_tmp_founds, cap * sizeof(bool), stream));
  CUDA_CHECK(hipMemsetAsync(d_tmp_founds, 0, cap * sizeof(bool), stream));
  h_tmp_keys = (K*)malloc(cap * sizeof(K));
  h_tmp_values = (V*)malloc(cap * dim * sizeof(V));
  h_tmp_scores = (S*)malloc(cap * sizeof(S));
  h_tmp_founds = (bool*)malloc(cap * sizeof(bool));

  size_t table_size_verify0 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_scores, stream);
  ASSERT_EQ(table_size_before, table_size_verify0);

  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_before * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_before * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores, d_tmp_scores,
                             table_size_before * sizeof(S),
                             hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys + table_size_before, keys,
                             len * sizeof(K), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values + table_size_before * dim, values,
                             len * dim * sizeof(V), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores + table_size_before, scores,
                             len * sizeof(S), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  for (size_t i = 0; i < cap; i++) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    values_map_before_insert[h_tmp_keys[i]] = *vec;
  }

  for (size_t i = 0; i < table_size_before; i++) {
    scores_map_before_insert[h_tmp_keys[i]] = h_tmp_scores[i];
  }

  table->assign_values(len, keys, values, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  {
    size_t table_size_verify1 = table->export_batch(
        table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_scores, stream);

    CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                               table_size_before * sizeof(K),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                               table_size_before * dim * sizeof(V),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_tmp_scores, d_tmp_scores,
                               table_size_before * sizeof(S),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_EQ(table_size_verify1, table_size_before);

    size_t score_error_cnt = 0;

    for (int64_t i = table_size_before - 1; i >= 0; i--) {
      test_util::ValueArray<V, dim>* vec =
          reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                           i * dim);
      values_map_after_insert[h_tmp_keys[i]] = *vec;
      scores_map_after_insert[h_tmp_keys[i]] = h_tmp_scores[i];
    }

    for (auto it : scores_map_current_batch) {
      const K key = it.first;
      const K score = it.second;
      S current_score = scores_map_after_insert[key];
      S score_before_insert = 0;
      if (scores_map_before_insert.find(key) !=
          scores_map_before_insert.end()) {
        score_before_insert = scores_map_before_insert[key];
        bool valid = ((current_score >> 32) < global_epoch) &&
                     ((current_score & 0xFFFFFFFF) ==
                      (0xFFFFFFFF & score_before_insert));

        if (!valid) {
          score_error_cnt++;
        }
      }
    }
    std::cout << "Check assign behavior got "
              << ", score_error_cnt: " << score_error_cnt
              << ", while len: " << len << std::endl;
    ASSERT_EQ(score_error_cnt, 0);
  }

  for (int64_t i = 0; i < table_size_before; i++) {
    values_map_before_insert[h_tmp_keys[i]] =
        values_map_after_insert[h_tmp_keys[i]];
    scores_map_before_insert[h_tmp_keys[i]] =
        scores_map_after_insert[h_tmp_keys[i]];
  }
  values_map_after_insert.clear();
  scores_map_after_insert.clear();

  EvictStrategy::set_global_epoch(global_epoch);
  auto start = std::chrono::steady_clock::now();
  size_t filtered_len = table->insert_and_evict(
      len, keys, values,
      (Table::evict_strategy == EvictStrategy::kLru ||
       Table::evict_strategy == EvictStrategy::kEpochLru)
          ? nullptr
          : scores,
      evicted_keys, evicted_values, evicted_scores, stream);
  evict_buffer->SyncData(false, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  auto end = std::chrono::steady_clock::now();
  auto diff = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

  for (size_t i = 0; i < filtered_len; i++) {
    scores_map_current_evict[evict_buffer->keys_ptr(false)[i]] =
        evict_buffer->scores_ptr(false)[i];
  }

  float dur = diff.count();

  size_t table_size_after = table->size(stream);
  size_t table_size_verify1 = table->export_batch(
      table->capacity(), 0, d_tmp_keys, d_tmp_values, d_tmp_scores, stream);

  ASSERT_EQ(table_size_verify1, table_size_after);

  size_t new_cap = table_size_after + filtered_len;
  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys, d_tmp_keys,
                             table_size_after * sizeof(K),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values, d_tmp_values,
                             table_size_after * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores, d_tmp_scores,
                             table_size_after * sizeof(S),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_keys + table_size_after, evicted_keys,
                             filtered_len * sizeof(K), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_values + table_size_after * dim,
                             evicted_values, filtered_len * dim * sizeof(V),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipMemcpyAsync(h_tmp_scores + table_size_after, evicted_scores,
                             filtered_len * sizeof(S), hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  size_t key_miss_cnt = 0;
  size_t value_diff_cnt = 0;
  size_t score_error_cnt1 = 0;
  size_t score_error_cnt2 = 0;

  for (int64_t i = new_cap - 1; i >= 0; i--) {
    test_util::ValueArray<V, dim>* vec =
        reinterpret_cast<test_util::ValueArray<V, dim>*>(h_tmp_values +
                                                         i * dim);
    values_map_after_insert[h_tmp_keys[i]] = *vec;
    scores_map_after_insert[h_tmp_keys[i]] = h_tmp_scores[i];
    if (i >= (new_cap - filtered_len)) {
      bool valid = ((h_tmp_scores[i] >> 32) < (global_epoch - 2));
      if (!valid) {
        score_error_cnt1++;
      }
    }
  }

  for (auto it : scores_map_current_batch) {
    const K key = it.first;
    const K score = it.second;
    S current_score = scores_map_after_insert[key];
    S score_before_insert = 0;
    if (values_map_after_insert.find(key) != values_map_after_insert.end() &&
        scores_map_current_evict.find(key) == scores_map_current_evict.end()) {
      score_before_insert = scores_map_before_insert[key];
    }
    bool valid = ((current_score >> 32) == global_epoch) &&
                 ((current_score & 0xFFFFFFFF) ==
                  ((0xFFFFFFFF & score_before_insert) + (0xFFFFFFFF & score)));

    if (!valid) {
      score_error_cnt2++;
    }
  }

  for (auto& it : values_map_before_insert) {
    if (values_map_after_insert.find(it.first) ==
        values_map_after_insert.end()) {
      ++key_miss_cnt;
      continue;
    }
    test_util::ValueArray<V, dim>& vec0 = it.second;
    test_util::ValueArray<V, dim>& vec1 = values_map_after_insert.at(it.first);
    for (size_t j = 0; j < dim; j++) {
      if (vec0[j] != vec1[j]) {
        ++value_diff_cnt;
        break;
      }
    }
  }

  std::cout << "Check insert_and_evict behavior got "
            << "key_miss_cnt: " << key_miss_cnt
            << ", value_diff_cnt: " << value_diff_cnt
            << ", score_error_cnt1: " << score_error_cnt1
            << ", score_error_cnt2: " << score_error_cnt2
            << ", while table_size_before: " << table_size_before
            << ", while table_size_after: " << table_size_after
            << ", while len: " << len << std::endl;

  ASSERT_EQ(key_miss_cnt, 0);
  ASSERT_EQ(value_diff_cnt, 0);
  ASSERT_EQ(score_error_cnt1, 0);
  ASSERT_EQ(score_error_cnt2, 0);

  CUDA_CHECK(hipFreeAsync(d_tmp_keys, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_values, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_scores, stream));
  CUDA_CHECK(hipFreeAsync(d_tmp_founds, stream));
  free(h_tmp_keys);
  free(h_tmp_values);
  free(h_tmp_scores);
  free(h_tmp_founds);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

void test_assign_advanced_on_epochlfu(size_t max_hbm_for_vectors) {
  const size_t U = 1024 * 1024;
  const size_t B = 100000;
  constexpr size_t dim = 16;

  TableOptions opt;

  opt.max_capacity = U;
  opt.init_capacity = U;
  opt.max_hbm_for_vectors = U * dim * sizeof(V);
  opt.max_bucket_size = 128;
  opt.max_hbm_for_vectors = nv::merlin::GB(max_hbm_for_vectors);
  using Table = nv::merlin::HashTable<K, V, S, EvictStrategy::kEpochLfu>;
  opt.dim = dim;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  std::unique_ptr<Table> table = std::make_unique<Table>();
  table->init(opt);

  test_util::KVMSBuffer<K, V, S> evict_buffer;
  evict_buffer.Reserve(B, dim, stream);
  evict_buffer.ToZeros(stream);

  test_util::KVMSBuffer<K, V, S> data_buffer;
  test_util::KVMSBuffer<K, V, S> pre_data_buffer;
  data_buffer.Reserve(B, dim, stream);
  pre_data_buffer.Reserve(B, dim, stream);

  size_t offset = 0;
  int freq_range = 100;
  float repeat_rate = 0.9;
  for (unsigned int global_epoch = 1; global_epoch <= 20; global_epoch++) {
    repeat_rate = global_epoch <= 1 ? 0.0 : 0.1;
    if (global_epoch <= 1) {
      test_util::create_random_keys_advanced<K, S, V>(
          dim, data_buffer.keys_ptr(false), data_buffer.scores_ptr(false),
          data_buffer.values_ptr(false), (int)B, B * 32, freq_range);
    } else {
      test_util::create_random_keys_advanced<K, S, V>(
          dim, data_buffer.keys_ptr(false), pre_data_buffer.keys_ptr(false),
          data_buffer.scores_ptr(false), data_buffer.values_ptr(false), (int)B,
          B * 32, freq_range, repeat_rate);
    }
    data_buffer.SyncData(true, stream);
    if (global_epoch <= 1) {
      pre_data_buffer.CopyFrom(data_buffer, stream);
    }

    CheckAssignOnEpochLfu<K, V, S, Table, dim>(table.get(), &data_buffer,
                                               &evict_buffer, &pre_data_buffer,
                                               B, stream, opt, global_epoch);

    pre_data_buffer.CopyFrom(data_buffer, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    offset += B;
  }
}

TEST(AssignValuesTest, test_evict_strategy_lru_basic) {
  test_evict_strategy_lru_basic(16);
  test_evict_strategy_lru_basic(0);
}
TEST(AssignValuesTest, test_evict_strategy_epochlfu_basic) {
  test_evict_strategy_epochlfu_basic(16);
  test_evict_strategy_epochlfu_basic(0);
}
TEST(AssignValuesTest, test_assign_advanced_on_epochlfu) {
  test_assign_advanced_on_epochlfu(16);
}